#include "hip/hip_runtime.h"
/*
 * Copyright (C)  NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
#include <math.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
//---
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
//---
#include <likwid-marker.h>

#include <device.h>

extern "C" {
#include <allocate.h>
#include <atom.h>
#include <comm.h>
#include <force.h>
#include <neighbor.h>
#include <parameter.h>
#include <timing.h>
#include <util.h>
}

__global__ void computeForceLJCudaFullNeigh(DeviceAtom a,
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon,
    int Nlocal,
    int neigh_maxneighs,
    int* neigh_neighbors,
    int* neigh_numneigh,
    int ntypes) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= Nlocal) {
        return;
    }

    DeviceAtom* atom    = &a;
    const int numneighs = neigh_numneigh[i];

    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);

    MD_FLOAT fix = 0;
    MD_FLOAT fiy = 0;
    MD_FLOAT fiz = 0;

#ifndef ONE_ATOM_TYPE
    const int type_i = atom->type[i];
#endif

    for (int k = 0; k < numneighs; k++) {
        int j         = neigh_neighbors[Nlocal * k + i];
        MD_FLOAT delx = xtmp - atom_x(j);
        MD_FLOAT dely = ytmp - atom_y(j);
        MD_FLOAT delz = ztmp - atom_z(j);
        MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

#ifndef ONE_ATOM_TYPE
        const int type_j          = atom->type[j];
        const int type_ij         = type_i * ntypes + type_j;
        const MD_FLOAT cutforcesq = atom->cutforcesq[type_ij];
        const MD_FLOAT sigma6     = atom->sigma6[type_ij];
        const MD_FLOAT epsilon    = atom->epsilon[type_ij];
#endif

        if (rsq < cutforcesq) {
            MD_FLOAT sr2   = (MD_FLOAT)1.0 / rsq;
            MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
            MD_FLOAT force = (MD_FLOAT)48.0 * sr6 * (sr6 - (MD_FLOAT)0.5) * sr2 * epsilon;
            fix += delx * force;
            fiy += dely * force;
            fiz += delz * force;
        }
    }

    atom_fx(i) = fix;
    atom_fy(i) = fiy;
    atom_fz(i) = fiz;
}

__global__ void computeForceLJCudaHalfNeigh(DeviceAtom a,
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon,
    int Nlocal,
    int neigh_maxneighs,
    int* neigh_neighbors,
    int* neigh_numneigh,
    int ntypes) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= Nlocal) {
        return;
    }

    DeviceAtom* atom    = &a;
    const int numneighs = neigh_numneigh[i];

    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);

    MD_FLOAT fix = 0;
    MD_FLOAT fiy = 0;
    MD_FLOAT fiz = 0;

#ifndef ONE_ATOM_TYPE
    const int type_i = atom->type[i];
#endif

    for (int k = 0; k < numneighs; k++) {
        int j         = neigh_neighbors[Nlocal * k + i];
        MD_FLOAT delx = xtmp - atom_x(j);
        MD_FLOAT dely = ytmp - atom_y(j);
        MD_FLOAT delz = ztmp - atom_z(j);
        MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

#ifndef ONE_ATOM_TYPE
        const int type_j          = atom->type[j];
        const int type_ij         = type_i * ntypes + type_j;
        const MD_FLOAT cutforcesq = atom->cutforcesq[type_ij];
        const MD_FLOAT sigma6     = atom->sigma6[type_ij];
        const MD_FLOAT epsilon    = atom->epsilon[type_ij];
#endif

        if (rsq < cutforcesq) {
            MD_FLOAT sr2   = (MD_FLOAT)1.0 / rsq;
            MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
            MD_FLOAT force = (MD_FLOAT)48.0 * sr6 * (sr6 - (MD_FLOAT)0.5) * sr2 * epsilon;
            MD_FLOAT partial_force_x = delx * force;
            MD_FLOAT partial_force_y = dely * force;
            MD_FLOAT partial_force_z = delz * force;

            atomicAdd(&atom_fx(j), -partial_force_x);
            atomicAdd(&atom_fy(j), -partial_force_y);
            atomicAdd(&atom_fz(j), -partial_force_z);

            fix += partial_force_x;
            fiy += partial_force_y;
            fiz += partial_force_z;
        }
    }

    atomicAdd(&atom_fx(i), fix);
    atomicAdd(&atom_fy(i), fiy);
    atomicAdd(&atom_fz(i), fiz);
}

__global__ void kernel_initial_integrate(
    MD_FLOAT dtforce, MD_FLOAT dt, int Nlocal, DeviceAtom a) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= Nlocal) {
        return;
    }

    DeviceAtom* atom = &a;

    atom_vx(i) += dtforce * atom_fx(i);
    atom_vy(i) += dtforce * atom_fy(i);
    atom_vz(i) += dtforce * atom_fz(i);
    atom_x(i) = atom_x(i) + dt * atom_vx(i);
    atom_y(i) = atom_y(i) + dt * atom_vy(i);
    atom_z(i) = atom_z(i) + dt * atom_vz(i);
}

__global__ void kernel_final_integrate(MD_FLOAT dtforce, int Nlocal, DeviceAtom a) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= Nlocal) {
        return;
    }

    DeviceAtom* atom = &a;

    atom_vx(i) += dtforce * atom_fx(i);
    atom_vy(i) += dtforce * atom_fy(i);
    atom_vz(i) += dtforce * atom_fz(i);
}

extern "C" {

void finalIntegrateCUDA(bool reneigh, Parameter* param, Atom* atom) {
    const int Nlocal                = atom->Nlocal;
    const int num_threads_per_block = get_cuda_num_threads();
    const int num_blocks            = ceil((float)Nlocal / (float)num_threads_per_block);

    kernel_final_integrate<<<num_blocks, num_threads_per_block>>>(param->dtforce,
        Nlocal,
        atom->d_atom);
    cuda_assert("kernel_final_integrate", hipPeekAtLastError());
    cuda_assert("kernel_final_integrate", hipDeviceSynchronize());

    if (reneigh) {
        memcpyFromGPU(atom->vx, atom->d_atom.vx, sizeof(MD_FLOAT) * atom->Nlocal * 3);
    }
}

void initialIntegrateCUDA(bool reneigh, Parameter* param, Atom* atom) {
    const int Nlocal                = atom->Nlocal;
    const int num_threads_per_block = get_cuda_num_threads();
    const int num_blocks            = ceil((float)Nlocal / (float)num_threads_per_block);

    kernel_initial_integrate<<<num_blocks, num_threads_per_block>>>(param->dtforce,
        param->dt,
        Nlocal,
        atom->d_atom);
    cuda_assert("kernel_initial_integrate", hipPeekAtLastError());
    cuda_assert("kernel_initial_integrate", hipDeviceSynchronize());

    if (reneigh) {
        memcpyFromGPU(atom->vx, atom->d_atom.vx, sizeof(MD_FLOAT) * atom->Nlocal * 3);
    }
}

double computeForceLJCUDA(Parameter* param, Atom* atom, Neighbor* neighbor, Stats* stats) {
    const int num_threads_per_block = get_cuda_num_threads();
    int Nlocal                      = atom->Nlocal;
    int Nmax                        = atom->Nmax;
    MD_FLOAT cutforcesq             = param->cutforce * param->cutforce;
    MD_FLOAT sigma6                 = param->sigma6;
    MD_FLOAT epsilon                = param->epsilon;

    /*
    int nDevices;
    hipGetDeviceCount(&nDevices);
    size_t free, total;
    for(int i = 0; i < nDevices; ++i) {
        hipMemGetInfo( &free, &total );
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("DEVICE %d/%d NAME: %s\r\n with %ld MB/%ld MB memory used", i + 1,
    nDevices, prop.name, free / 1024 / 1024, total / 1024 / 1024);
    }
    */

    // HINT: Run with cuda-memcheck ./MDBench-NVCC in case of error
    // memsetGPU(atom->d_atom.fx, 0, sizeof(MD_FLOAT) * Nlocal * 3);

    hipProfilerStart();
    const int num_blocks = ceil((float)Nlocal / (float)num_threads_per_block);
    double S             = getTimeStamp();
    LIKWID_MARKER_START("force");

    if (neighbor->half_neigh) {
#ifdef AOS
        memsetGPU(atom->d_atom.fx, 0, sizeof(MD_FLOAT) * Nmax * 3);
#else
        memsetGPU(atom->d_atom.fx, 0, sizeof(MD_FLOAT) * Nmax);
        memsetGPU(atom->d_atom.fy, 0, sizeof(MD_FLOAT) * Nmax);
        memsetGPU(atom->d_atom.fz, 0, sizeof(MD_FLOAT) * Nmax);
#endif
        computeForceLJCudaHalfNeigh<<<num_blocks, num_threads_per_block>>>(atom->d_atom,
            cutforcesq,
            sigma6,
            epsilon,
            Nlocal,
            neighbor->maxneighs,
            neighbor->d_neighbor.neighbors,
            neighbor->d_neighbor.numneigh,
            atom->ntypes);
    } else {
        computeForceLJCudaFullNeigh<<<num_blocks, num_threads_per_block>>>(atom->d_atom,
            cutforcesq,
            sigma6,
            epsilon,
            Nlocal,
            neighbor->maxneighs,
            neighbor->d_neighbor.neighbors,
            neighbor->d_neighbor.numneigh,
            atom->ntypes);
    }

    cuda_assert("computeForceLJCuda", hipPeekAtLastError());
    cuda_assert("computeForceLJCuda", hipDeviceSynchronize());
    hipProfilerStop();

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    return E - S;
}
}

extern "C" void copyGhostFromGPU(Atom* atom) {
    memcpyFromGPU(atom->x, atom->d_atom.x, atom->Nlocal * sizeof(MD_FLOAT) * 3);
}

extern "C" void copyGhostToGPU(Atom* atom) {
    memcpyToGPU(&atom->d_atom.x[atom->Nlocal * 3],
        &atom->x[atom->Nlocal * 3],
        atom->Nghost * sizeof(MD_FLOAT) * 3);
}

extern "C" void copyForceFromGPU(Atom* atom) {
    memcpyFromGPU(atom->fx, atom->d_atom.fx, atom->Nmax * sizeof(MD_FLOAT) * 3);
}

extern "C" void copyForceToGPU(Atom* atom) {
    memcpyToGPU(atom->d_atom.fx, atom->fx, atom->Nmax * sizeof(MD_FLOAT) * 3);
}

extern "C" void copyDataFromCUDADevice(Parameter* param, Atom* atom) {
    memcpyFromGPU(atom->x, atom->d_atom.x, atom->Nmax * sizeof(MD_FLOAT) * 3);
    memcpyFromGPU(atom->vx, atom->d_atom.vx, atom->Nmax * sizeof(MD_FLOAT) * 3);
    memcpyFromGPU(atom->type, atom->d_atom.type, atom->Nmax * sizeof(int));
}

extern "C" void copyDataToCUDADevice(Parameter* param, Atom* atom) {
    memcpyToGPU(atom->d_atom.x, atom->x, atom->Nmax * sizeof(MD_FLOAT) * 3);
    memcpyToGPU(atom->d_atom.vx, atom->vx, atom->Nmax * sizeof(MD_FLOAT) * 3);
    memcpyToGPU(atom->d_atom.type, atom->type, atom->Nmax * sizeof(int));
}

/*
__global__ void pack_forward_Cuda(DeviceAtom a,
    int n,
    int* cuda_list,
    MD_FLOAT* buf,
    int PBCx,
    int PBCy,
    int PBCz,
    MD_FLOAT xprd,
    MD_FLOAT yprd,
    MD_FLOAT zprd)
{
    DeviceAtom* atom    = &a;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;
    int j        = cuda_list[i];
    buf_x(i) = atom_x(j) + PBCx * xprd;
    buf_y(i) = atom_y(j) + PBCy * yprd;
    buf_z(i) = atom_z(j) + PBCz * zprd;
}

__global__ void unpack_forward_Cuda(DeviceAtom a,
    int n,
    int first,
    MD_FLOAT* buf)
{
    DeviceAtom* atom    = &a;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;
    atom_x((first + i)) = buf_x(i);
    atom_y((first + i)) = buf_y(i);
    atom_z((first + i)) = buf_z(i);
}

extern "C" void forwardCommCUDA(Comm* comm, Atom* atom, int iswap)
{
    int nrqst = 0, offset = 0, nsend = 0, nrecv = 0;
    int pbc[3];
    int size    = comm->forwardSize;
    int maxrqst = comm->numneigh;
    int* cuda_sendlist;
    int max_list_size;
    cuda_buf_send =  (MD_FLOAT*)allocateGPU(comm->maxsend * sizeof(MD_FLOAT));
    cuda_buf_recv =  (MD_FLOAT*)allocateGPU(comm->maxrecv * sizeof(MD_FLOAT));

    //use a single buffer and takes the highes list size to move list of cluster to send
    for (int ineigh = 0; ineigh < comm->numneigh; ineigh++){
        max_list_size = comm->maxsendlist[ineigh];
    }
    //allocate the memory for the unique buffer
    cuda_sendlist = (int*)allocateGPU(max_list_size * sizeof(int));

    for (int ineigh = comm->sendfrom[iswap]; ineigh < comm->sendtill[iswap]; ineigh++) {
        offset  = comm->off_atom_send[ineigh];
        pbc[_x] = comm->pbc_x[ineigh];
        pbc[_y] = comm->pbc_y[ineigh];
        pbc[_z] = comm->pbc_z[ineigh];
        //copy lists into the buffer
        memcpyToGPU(cuda_sendlist, comm->sendlist[ineigh], comm->atom_send[ineigh] *
sizeof(int));

        const int num_threads_per_block = get_cuda_num_threads();
        const int num_blocks = ceil((float) comm->atom_send[ineigh] /
(float)num_threads_per_block);

        pack_forward_Cuda<<<num_blocks, num_threads_per_block>>>(
                                            atom->d_atom,                   //MD_FLOAT*
-->need to be in tye device comm->atom_send[ineigh],        //int cuda_sendlist, //int*
-->need to be in tye device &cuda_buf_send[offset*size],    //MD_FLOAT*  -->need to be in
tye device pbc[_x],                        //int pbc[_y],                        //int
                                            pbc[_z],                        //int
                                            atom->mybox.xprd, //MD_FLOAT atom->mybox.yprd,
//MD_FLOAT atom->mybox.zprd);                    //MD_FLOAT hipDeviceSynchronize();
        }

#ifdef _MPI
    MPI_Request requests[maxrqst];
    // Receives elements
    if (comm->othersend[iswap])
        for (int ineigh = comm->recvfrom[iswap]; ineigh < comm->recvtill[iswap];ineigh++)
{ offset = comm->off_atom_recv[ineigh] * size; nrecv  = comm->atom_recv[ineigh] * size;
            MPI_Irecv(&cuda_buf_recv[offset],
                    nrecv,
                    type,
                    comm->nrecv[ineigh],
                    0,
                    world,
                    &requests[nrqst++]);
        }

    // Send elements
    if (comm->othersend[iswap])
        for (int ineigh = comm->sendfrom[iswap]; ineigh < comm->sendtill[iswap]; ineigh++)
{ offset = comm->off_atom_send[ineigh] * size; nsend  = comm->atom_send[ineigh] * size;
            MPI_Send(&cuda_buf_send[offset], nsend, type, comm->nsend[ineigh], 0, world);
        }

    if (comm->othersend[iswap]) MPI_Waitall(nrqst, requests, MPI_STATUS_IGNORE);
#endif

    // unpack buffer
    for (int ineigh = comm->recvfrom[iswap]; ineigh < comm->recvtill[iswap]; ineigh++) {
        offset = comm->off_atom_recv[ineigh];
        MD_FLOAT *buf = (comm->othersend[iswap]) ? cuda_buf_recv : cuda_buf_send;

        const int num_threads_per_block = get_cuda_num_threads();
        const int num_blocks = ceil((float) comm->atom_send[ineigh] /
(float)num_threads_per_block);

        unpack_forward_Cuda<<<num_blocks, num_threads_per_block>>>(
                                        atom->d_atom,                       //MD_FLOAT*
--> need to be in the device comm->atom_recv[ineigh],            //int
                                        comm->firstrecv[iswap] + offset,    //int
                                        &buf[offset * size]);               //MD_FLOAT*
--> need to be in the devic hipDeviceSynchronize();
        }
    cuda_assert("cudaDeviceFree", hipFree(cuda_sendlist));
    cuda_assert("cudaDeviceFree", hipFree(cuda_buf_recv));
    cuda_assert("cudaDeviceFree", hipFree(cuda_buf_send));
}
*/
