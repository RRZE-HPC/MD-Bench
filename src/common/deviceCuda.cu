/*
 * Copyright (C)  NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
 #include <stdio.h>
 #include <stdlib.h>
 //---
 #include <hip/hip_runtime.h>
 
 #include <device.h>

void cuda_assert(const char* label, hipError_t err)
{
    if (err != hipSuccess) {
        printf("[CUDA Error]: %s: %s\r\n", label, hipGetErrorString(err));
        exit(-1);
    }
}

void GPUfree(void * any) {
    cuda_assert("GPUfree", hipFree(any));
}

void* allocateGPU(size_t bytesize)
{
    void* ptr;
#ifdef CUDA_HOST_MEMORY
    cuda_assert("allocateGPU", hipHostMalloc((void**)&ptr, bytesize));
#else
    cuda_assert("allocateGPU", hipMalloc((void**)&ptr, bytesize));
#endif
    return ptr;
}

// Data is not preserved
void* reallocateGPU(void* ptr, size_t new_bytesize)
{
    if (ptr != NULL) {
#ifdef CUDA_HOST_MEMORY
        hipHostFree(ptr);
#else
        hipFree(ptr);
#endif
    }
    return allocateGPU(new_bytesize);
}

void memcpyToGPU(void* d_ptr, void* h_ptr, size_t bytesize)
{
#ifndef CUDA_HOST_MEMORY
    cuda_assert("memcpyToGPU",
        hipMemcpy(d_ptr, h_ptr, bytesize, hipMemcpyHostToDevice));
#endif
}

void memcpyFromGPU(void* h_ptr, void* d_ptr, size_t bytesize)
{
#ifndef CUDA_HOST_MEMORY
    cuda_assert("memcpyFromGPU",
        hipMemcpy(h_ptr, d_ptr, bytesize, hipMemcpyDeviceToHost));
#endif
}

void memsetGPU(void* d_ptr, int value, size_t bytesize)
{
    cuda_assert("memsetGPU", hipMemset(d_ptr, value, bytesize));
}