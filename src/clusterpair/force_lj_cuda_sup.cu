#include "hip/hip_runtime.h"
extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <force.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>
}

extern "C" {
extern MD_FLOAT* cuda_cl_x;
extern MD_FLOAT* cuda_cl_v;
extern MD_FLOAT* cuda_cl_f;
extern int* cuda_neighbors;
extern int* cuda_numneigh;
extern int* cuda_natoms;
extern int* natoms;
extern int* ngatoms;
extern int* cuda_border_map;
extern int* cuda_jclusters_natoms;
extern MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
extern MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
extern MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
extern int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
}

__global__ void cudaInitialIntegrateSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    MD_FLOAT dtforce,
    MD_FLOAT dt) {

    int sci = blockDim.x * blockIdx.x + threadIdx.x;
    int ci = threadIdx.y;
    int cii = threadIdx.z;

    if (sci >= Nclusters_local) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    int i            = ci * CLUSTER_M + cii;
    MD_FLOAT* ci_x   = &cuda_cl_x[sci_vec_base];
    MD_FLOAT* ci_v   = &cuda_cl_v[sci_vec_base];
    MD_FLOAT* ci_f   = &cuda_cl_f[sci_vec_base];

    ci_v[CL_X_OFFSET + i] += dtforce * ci_f[CL_X_OFFSET + i];
    ci_v[CL_Y_OFFSET + i] += dtforce * ci_f[CL_Y_OFFSET + i];
    ci_v[CL_Z_OFFSET + i] += dtforce * ci_f[CL_Z_OFFSET + i];
    ci_x[CL_X_OFFSET + i] += dt * ci_v[CL_X_OFFSET + i];
    ci_x[CL_Y_OFFSET + i] += dt * ci_v[CL_Y_OFFSET + i];
    ci_x[CL_Z_OFFSET + i] += dt * ci_v[CL_Z_OFFSET + i];
}

__global__ void cudaFinalIntegrateSup_warp(MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    MD_FLOAT dtforce) {

    int sci = blockDim.x * blockIdx.x + threadIdx.x;
    int ci = threadIdx.y;
    int cii = threadIdx.z;

    if (sci >= Nclusters_local) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    int i            = ci * CLUSTER_M + cii;
    MD_FLOAT* ci_v   = &cuda_cl_v[sci_vec_base];
    MD_FLOAT* ci_f   = &cuda_cl_f[sci_vec_base];

    ci_v[CL_X_OFFSET + i] += dtforce * ci_f[CL_X_OFFSET + i];
    ci_v[CL_Y_OFFSET + i] += dtforce * ci_f[CL_Y_OFFSET + i];
    ci_v[CL_Z_OFFSET + i] += dtforce * ci_f[CL_Z_OFFSET + i];
}

__global__ void computeForceLJCudaSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    int* cuda_numneigh,
    int* cuda_neighs,
    int half_neigh,
    int maxneighs,
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon) {

    int sci = blockIdx.x;
    int cii = threadIdx.x;
    int cjj = threadIdx.y;
    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    MD_FLOAT* sci_x  = &cuda_cl_x[sci_vec_base];
    MD_FLOAT* sci_f  = &cuda_cl_f[sci_vec_base];

    for(int k = 0; k < cuda_numneigh[sci]; k++) {
        int cj = cuda_neighs[sci * maxneighs + k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT* cj_x  = &cuda_cl_x[cj_vec_base];
        MD_FLOAT* cj_f  = &cuda_cl_f[cj_vec_base];
        MD_FLOAT xjtmp = cj_x[CL_X_OFFSET + cjj];
        MD_FLOAT yjtmp = cj_x[CL_Y_OFFSET + cjj];
        MD_FLOAT zjtmp = cj_x[CL_Z_OFFSET + cjj];

        for(int ci = 0; ci < SCLUSTER_SIZE; ci++) {
            if(sci != cj / SCLUSTER_SIZE || ci != cj % SCLUSTER_SIZE || cii != cjj) {
                MD_FLOAT delx = sci_x[CL_X_OFFSET + ci * CLUSTER_M + cii] - xjtmp;
                MD_FLOAT dely = sci_x[CL_Y_OFFSET + ci * CLUSTER_M + cii] - yjtmp;
                MD_FLOAT delz = sci_x[CL_Z_OFFSET + ci * CLUSTER_M + cii] - zjtmp;
                MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

                if(rsq < cutforcesq) {
                    MD_FLOAT sr2   = (MD_FLOAT)1.0 / rsq;
                    MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
                    MD_FLOAT force = (MD_FLOAT)48.0 * sr6 * (sr6 - (MD_FLOAT)0.5) * sr2 * epsilon;

                    if (half_neigh) {
                        atomicAdd(&cj_f[CL_X_OFFSET + cjj], -delx * force);
                        atomicAdd(&cj_f[CL_Y_OFFSET + cjj], -dely * force);
                        atomicAdd(&cj_f[CL_Z_OFFSET + cjj], -delz * force);
                    }

                    atomicAdd(&sci_f[CL_X_OFFSET + ci * CLUSTER_M + cii], delx * force);
                    atomicAdd(&sci_f[CL_Y_OFFSET + ci * CLUSTER_M + cii], dely * force);
                    atomicAdd(&sci_f[CL_Z_OFFSET + ci * CLUSTER_M + cii], delz * force);
                }

                if(fabs(rsq) < 1e-6) {
                    printf(
                        "Distance rsq close to zero: sci=%d/%d, cj=%d/%d, cii=%d cjj=%d rsq=%e\n",
                        sci, Nclusters_local, cj, Nclusters_local * SCLUSTER_SIZE, cii, cjj, rsq);
                    printf(
                        "i: %d, j: %d\n",
                        sci_vec_base + ci * CLUSTER_M + cii, cj_vec_base + cjj);
                    printf(
                        "Positions: <%.4f, %.4f, %.4f>, <%.4f, %.4f, %.4f>\n",
                        sci_x[CL_X_OFFSET + ci * CLUSTER_M + cii],
                        sci_x[CL_Y_OFFSET + ci * CLUSTER_M + cii],
                        sci_x[CL_Z_OFFSET + ci * CLUSTER_M + cii],
                        xjtmp, yjtmp, zjtmp);
                }
            }
        }
    }
}

__global__ void cudaUpdatePbcSup_warp(MD_FLOAT* cuda_cl_x,
    int* cuda_border_map,
    int* cuda_jclusters_natoms,
    int* cuda_PBCx,
    int* cuda_PBCy,
    int* cuda_PBCz,
    int Nclusters_local,
    int Nclusters_ghost,
    MD_FLOAT param_xprd,
    MD_FLOAT param_yprd,
    MD_FLOAT param_zprd) {

    int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) {
        return;
    }

    int ncj             = Nclusters_local * SCLUSTER_SIZE;
    int cj              = ncj + cg;
    int cj_vec_base     = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base   = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT* cj_x      = &cuda_cl_x[cj_vec_base];
    MD_FLOAT* bmap_x    = &cuda_cl_x[bmap_vec_base];

    for (int cjj = 0; cjj < CLUSTER_N; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * param_xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * param_yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * param_zprd;
    }
}

extern "C" double computeForceLJCudaSup(Parameter* param, Atom* atom, Neighbor* neighbor, Stats* stats) {
    DEBUG_MESSAGE("computeForceLJCudaSup start\r\n");

    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6     = param->sigma6;
    MD_FLOAT epsilon    = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * SCLUSTER_SIZE * 3 * sizeof(MD_FLOAT));
    dim3 block_size       = dim3(CLUSTER_M, CLUSTER_N, 1);
    dim3 grid_size        = dim3(atom->Nclusters_local, 1, 1);
    double S              = getTimeStamp();
    LIKWID_MARKER_START("force");

    computeForceLJCudaSup_warp<<<grid_size, block_size>>>(cuda_cl_x,
        cuda_cl_f,
        atom->Nclusters_local,
        cuda_numneigh,
        cuda_neighbors,
        neighbor->half_neigh,
        neighbor->maxneighs,
        cutforcesq,
        sigma6,
        epsilon);

    cuda_assert("computeForceLJCudaSup", hipPeekAtLastError());
    cuda_assert("computeForceLJCudaSup", hipDeviceSynchronize());

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    DEBUG_MESSAGE("computeForceLJCudaSup stop\r\n");
    return E - S;
}
