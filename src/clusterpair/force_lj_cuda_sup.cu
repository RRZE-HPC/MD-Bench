#include "hip/hip_runtime.h"
extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <force.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>
}

extern "C" {
extern MD_FLOAT* cuda_cl_x;
extern MD_FLOAT* cuda_cl_v;
extern MD_FLOAT* cuda_cl_f;
extern int* cuda_neighbors;
extern int* cuda_numneigh;
extern int* cuda_natoms;
extern int* natoms;
extern int* ngatoms;
extern int* cuda_border_map;
extern int* cuda_jclusters_natoms;
extern MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
extern MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
extern MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
extern int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
extern int isReneighboured;

extern int* cuda_iclusters;
extern int* cuda_nclusters;

extern MD_FLOAT* cuda_scl_x;
extern MD_FLOAT* cuda_scl_v;
extern MD_FLOAT* cuda_scl_f;
}

__global__ void cudaInitialIntegrateSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int* cuda_nclusters,
    int* cuda_natoms,
    int Nsclusters_local,
    MD_FLOAT dtforce,
    MD_FLOAT dt)
{

    unsigned int sci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    // unsigned int cii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    if (sci_pos >= Nsclusters_local) return;

    // unsigned int ci_pos = cii_pos / CLUSTER_M;
    // unsigned int scii_pos = cii_pos % CLUSTER_M;

    // if (ci_pos >= cuda_nclusters[sci_pos]) return;
    // if (scii_pos >= cuda_natoms[ci_pos]) return;

    int ci_vec_base = SCI_VECTOR_BASE_INDEX(sci_pos);
    MD_FLOAT* ci_x  = &cuda_cl_x[ci_vec_base];
    MD_FLOAT* ci_v  = &cuda_cl_v[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];

    for (int scii_pos = 0; scii_pos < SCLUSTER_M; scii_pos++) {
        ci_v[SCL_X_OFFSET + scii_pos] += dtforce * ci_f[SCL_X_OFFSET + scii_pos];
        ci_v[SCL_Y_OFFSET + scii_pos] += dtforce * ci_f[SCL_Y_OFFSET + scii_pos];
        ci_v[SCL_Z_OFFSET + scii_pos] += dtforce * ci_f[SCL_Z_OFFSET + scii_pos];
        ci_x[SCL_X_OFFSET + scii_pos] += dt * ci_v[SCL_X_OFFSET + scii_pos];
        ci_x[SCL_Y_OFFSET + scii_pos] += dt * ci_v[SCL_Y_OFFSET + scii_pos];
        ci_x[SCL_Z_OFFSET + scii_pos] += dt * ci_v[SCL_Z_OFFSET + scii_pos];
    }
}

__global__ void cudaFinalIntegrateSup_warp(MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int* cuda_nclusters,
    int* cuda_natoms,
    int Nsclusters_local,
    MD_FLOAT dtforce)
{

    unsigned int sci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    // unsigned int cii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    if (sci_pos >= Nsclusters_local) return;

    // unsigned int ci_pos = cii_pos / CLUSTER_M;
    // unsigned int scii_pos = cii_pos % CLUSTER_M;

    // if (ci_pos >= cuda_nclusters[sci_pos]) return;
    // if (scii_pos >= cuda_natoms[ci_pos]) return;

    int ci_vec_base = SCI_VECTOR_BASE_INDEX(sci_pos);
    MD_FLOAT* ci_v  = &cuda_cl_v[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];

    for (int scii_pos = 0; scii_pos < SCLUSTER_M; scii_pos++) {
        ci_v[SCL_X_OFFSET + scii_pos] += dtforce * ci_f[SCL_X_OFFSET + scii_pos];
        ci_v[SCL_Y_OFFSET + scii_pos] += dtforce * ci_f[SCL_Y_OFFSET + scii_pos];
        ci_v[SCL_Z_OFFSET + scii_pos] += dtforce * ci_f[SCL_Z_OFFSET + scii_pos];
    }
}

__global__ void computeForceLJCudaSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_f,
    int* cuda_nclusters,
    int* cuda_iclusters,
    int Nsclusters_local,
    int* cuda_numneigh,
    int* cuda_neighs,
    int half_neigh,
    int maxneighs,
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon) {

    unsigned int sci = blockIdx.x;
    //unsigned int ci = sci * SCLUSTER_SIZE + threadIdx.y;
    unsigned int cii = threadIdx.x;
    unsigned int cjj = threadIdx.y;

    if ((sci >= Nsclusters_local) || (cii >= SCLUSTER_M) || (cjj >= CLUSTER_N)) {
        return;
    }

    int ci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    MD_FLOAT* ci_x  = &cuda_cl_x[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];
    MD_FLOAT fix  = 0;
    MD_FLOAT fiy  = 0;
    MD_FLOAT fiz  = 0;

    for (int k = 0; k < cuda_numneigh[sci]; k++) {
        int cj = cuda_neighs[sci * maxneighs + k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(scj);
        MD_FLOAT* cj_x  = &cuda_cl_x[cj_vec_base];
        MD_FLOAT* cj_f  = &cuda_cl_f[cj_vec_base];
        MD_FLOAT xjtmp = cj_x[SCL_X_OFFSET + cjj];
        MD_FLOAT yjtmp = cj_x[SCL_Y_OFFSET + cjj];
        MD_FLOAT zjtmp = cj_x[SCL_Z_OFFSET + cjj];

        for(int ci = 0; ci < SCLUSTER_SIZE; ci++) {
            int cond = cj / SCLUSTER_SIZE != sci ||
                    threadIdx.y != cj % SCLUSTER_SIZE ||
                    threadIdx.x != cjj;

            if (cond) {
                MD_FLOAT delx = ci_x[SCL_X_OFFSET + cii] - xjtmp;
                MD_FLOAT dely = ci_x[SCL_Y_OFFSET + cii] - yjtmp;
                MD_FLOAT delz = ci_x[SCL_Z_OFFSET + cii] - zjtmp
                MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

                if (rsq < cutforcesq) {
                    MD_FLOAT sr2   = 1.0 / rsq;
                    MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
                    MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;

                    if (half_neigh) {
                        atomicAdd(&cj_f[SCL_X_OFFSET + cjj], -delx * force);
                        atomicAdd(&cj_f[SCL_Y_OFFSET + cjj], -dely * force);
                        atomicAdd(&cj_f[SCL_Z_OFFSET + cjj], -delz * force);
                    }

                    atomicAdd(&ci_f[SCL_X_OFFSET + cii], delx * force);
                    atomicAdd(&ci_f[SCL_Y_OFFSET + cii], dely * force);
                    atomicAdd(&ci_f[SCL_Z_OFFSET + cii], delz * force);
                }
            }
        }
    }
}

__global__ void cudaUpdatePbcSup_warp(MD_FLOAT* cuda_cl_x,
    int* cuda_border_map,
    int* cuda_jclusters_natoms,
    int* cuda_PBCx,
    int* cuda_PBCy,
    int* cuda_PBCz,
    int Nsclusters_local,
    int Nclusters_ghost,
    MD_FLOAT param_xprd,
    MD_FLOAT param_yprd,
    MD_FLOAT param_zprd)
{
    unsigned int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) return;

    // int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int jfac      = SCLUSTER_SIZE / CLUSTER_M;
    int ncj       = Nsclusters_local / jfac;
    MD_FLOAT xprd = param_xprd;
    MD_FLOAT yprd = param_yprd;
    MD_FLOAT zprd = param_zprd;

    const int cj      = ncj + cg;
    int cj_vec_base   = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT* cj_x    = &cuda_cl_x[cj_vec_base];
    MD_FLOAT* bmap_x  = &cuda_cl_x[bmap_vec_base];

    for (int cjj = 0; cjj < cuda_jclusters_natoms[cg]; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * zprd;
    }
}

extern "C" double computeForceLJCudaSup(
    Parameter* param, Atom* atom, Neighbor* neighbor, Stats* stats)
{
    DEBUG_MESSAGE("computeForceLJCudaSup start\r\n");

    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6     = param->sigma6;
    MD_FLOAT epsilon    = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    if (isReneighboured) {

        for (int ci = 0; ci < atom->Nclusters_local; ci++) {
            memcpyToGPU(&cuda_numneigh[ci], &neighbor->numneigh[ci], sizeof(int));
            memcpyToGPU(&cuda_neighbors[ci * neighbor->maxneighs],
                &neighbor->neighbors[ci * neighbor->maxneighs],
                neighbor->numneigh[ci] * sizeof(int));
        }

        for (int sci = 0; sci < atom->Nsclusters_local; sci++) {
            memcpyToGPU(&cuda_nclusters[sci],
                &atom->siclusters[sci].nclusters,
                sizeof(int));
            // memcpyToGPU(&cuda_iclusters[sci * SCLUSTER_SIZE],
            // &atom->siclusters[sci].iclusters, sizeof(int) *
            // atom->siclusters[sci].nclusters);
        }

        memcpyToGPU(cuda_iclusters,
            atom->icluster_idx,
            atom->Nsclusters_max * SCLUSTER_SIZE * sizeof(int));

        isReneighboured = 0;
    }

    const int threads_num = 1;
    dim3 block_size       = dim3(threads_num, SCLUSTER_M, CLUSTER_N);
    dim3 grid_size        = dim3(atom->Nsclusters_local / threads_num + 1, 1, 1);
    double S              = getTimeStamp();
    LIKWID_MARKER_START("force");

    computeForceLJCudaSup_warp<<<grid_size, block_size>>>(cuda_scl_x,
        cuda_scl_f,
        cuda_nclusters,
        cuda_iclusters,
        atom->Nsclusters_local,
        cuda_numneigh,
        cuda_neighbors,
        neighbor->half_neigh,
        neighbor->maxneighs,
        cutforcesq,
        sigma6,
        epsilon);
    cuda_assert("computeForceLJCudaSup", hipPeekAtLastError());
    cuda_assert("computeForceLJCudaSup", hipDeviceSynchronize());

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    DEBUG_MESSAGE("computeForceCudaSup stop\r\n");
    return E - S;
}
