#include "hip/hip_runtime.h"
extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <force.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>
}

extern "C" {
extern MD_FLOAT* cuda_cl_x;
extern MD_FLOAT* cuda_cl_v;
extern MD_FLOAT* cuda_cl_f;
extern int* cuda_neighbors;
extern int* cuda_numneigh;
extern int* cuda_natoms;
extern int* natoms;
extern int* ngatoms;
extern int* cuda_border_map;
extern int* cuda_jclusters_natoms;
extern MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
extern MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
extern MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
extern int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
}

__global__ void cudaInitialIntegrateSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    MD_FLOAT dtforce,
    MD_FLOAT dt) {

    int sci = blockIdx.x;
    int ci = threadIdx.x;
    int cii = threadIdx.y;

    if (sci >= Nclusters_local) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    int i            = ci * CLUSTER_M + cii;
    MD_FLOAT* ci_x   = &cuda_cl_x[sci_vec_base];
    MD_FLOAT* ci_v   = &cuda_cl_v[sci_vec_base];
    MD_FLOAT* ci_f   = &cuda_cl_f[sci_vec_base];

    ci_v[CL_X_OFFSET + i] += dtforce * ci_f[CL_X_OFFSET + i];
    ci_v[CL_Y_OFFSET + i] += dtforce * ci_f[CL_Y_OFFSET + i];
    ci_v[CL_Z_OFFSET + i] += dtforce * ci_f[CL_Z_OFFSET + i];
    ci_x[CL_X_OFFSET + i] += dt * ci_v[CL_X_OFFSET + i];
    ci_x[CL_Y_OFFSET + i] += dt * ci_v[CL_Y_OFFSET + i];
    ci_x[CL_Z_OFFSET + i] += dt * ci_v[CL_Z_OFFSET + i];
}

extern "C" void cudaInitialIntegrateSup(Parameter* param, Atom* atom) {
    dim3 block_size       = dim3(SCLUSTER_SIZE, CLUSTER_M, 1);
    dim3 grid_size        = dim3(atom->Nclusters_local, 1, 1);

    cudaInitialIntegrateSup_warp<<<grid_size, block_size>>>(cuda_cl_x,
        cuda_cl_v,
        cuda_cl_f,
        atom->Nclusters_local,
        param->dtforce,
        param->dt);

    cuda_assert("cudaInitialIntegrateSup", hipPeekAtLastError());
    cuda_assert("cudaInitialIntegrateSup", hipDeviceSynchronize());
}

__global__ void cudaFinalIntegrateSup_warp(MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    MD_FLOAT dtforce) {

    int sci = blockIdx.x;
    int ci = threadIdx.x;
    int cii = threadIdx.y;

    if (sci >= Nclusters_local) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    int i            = ci * CLUSTER_M + cii;
    MD_FLOAT* ci_v   = &cuda_cl_v[sci_vec_base];
    MD_FLOAT* ci_f   = &cuda_cl_f[sci_vec_base];

    ci_v[CL_X_OFFSET + i] += dtforce * ci_f[CL_X_OFFSET + i];
    ci_v[CL_Y_OFFSET + i] += dtforce * ci_f[CL_Y_OFFSET + i];
    ci_v[CL_Z_OFFSET + i] += dtforce * ci_f[CL_Z_OFFSET + i];
}

extern "C" void cudaFinalIntegrateSup(Parameter* param, Atom* atom) {
    dim3 block_size       = dim3(SCLUSTER_SIZE, CLUSTER_M, 1);
    dim3 grid_size        = dim3(atom->Nclusters_local, 1, 1);

    cudaFinalIntegrateSup_warp<<<grid_size, block_size>>>(cuda_cl_v,
        cuda_cl_f,
        atom->Nclusters_local,
        param->dt);

    cuda_assert("cudaFinalIntegrateSup", hipPeekAtLastError());
    cuda_assert("cudaFinalIntegrateSup", hipDeviceSynchronize());
}

__global__ void computeForceLJCudaSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    int* cuda_numneigh,
    int* cuda_neighs,
    int half_neigh,
    int maxneighs,
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon) {

    __shared__ MD_FLOAT sh_sci_x[SCLUSTER_SIZE * CLUSTER_M * 3];
    int sci = blockIdx.x;
    int cii = threadIdx.x;
    int cjj = threadIdx.y;
    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    MD_FLOAT* sci_x  = &cuda_cl_x[sci_vec_base];
    MD_FLOAT* sci_f  = &cuda_cl_f[sci_vec_base];
    int tid = cii * CLUSTER_N + cjj;
    int ncoords = SCLUSTER_SIZE * CLUSTER_M * 3;
    MD_FLOAT fx_acc[SCLUSTER_SIZE];
    MD_FLOAT fy_acc[SCLUSTER_SIZE];
    MD_FLOAT fz_acc[SCLUSTER_SIZE];

    #pragma unroll
    for(int i = 0; i < SCLUSTER_SIZE; i++) {
        fx_acc[i] = (MD_FLOAT)0.0;
        fy_acc[i] = (MD_FLOAT)0.0;
        fz_acc[i] = (MD_FLOAT)0.0;
    }

    for(int idx = tid; idx < ncoords; idx += blockDim.x * blockDim.y) {
        sh_sci_x[idx] = sci_x[idx];
    }

    __syncthreads();

    for(int k = 0; k < cuda_numneigh[sci]; k++) {
        int cj = cuda_neighs[sci * maxneighs + k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT* cj_x  = &cuda_cl_x[cj_vec_base];
        MD_FLOAT* cj_f  = &cuda_cl_f[cj_vec_base];
        MD_FLOAT xjtmp = cj_x[CL_X_OFFSET + cjj];
        MD_FLOAT yjtmp = cj_x[CL_Y_OFFSET + cjj];
        MD_FLOAT zjtmp = cj_x[CL_Z_OFFSET + cjj];
        int cj_sc = cj / SCLUSTER_SIZE;
        int sci_cj = cj % SCLUSTER_SIZE;

        #pragma unroll
        for(int sci_ci = 0; sci_ci < SCLUSTER_SIZE; sci_ci++) {
            if(sci != cj_sc || sci_ci != sci_cj || cii != cjj) {
                int ai = sci_ci * CLUSTER_M + cii;
                MD_FLOAT delx = sh_sci_x[CL_X_OFFSET + ai] - xjtmp;
                MD_FLOAT dely = sh_sci_x[CL_Y_OFFSET + ai] - yjtmp;
                MD_FLOAT delz = sh_sci_x[CL_Z_OFFSET + ai] - zjtmp;
                MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

                if(rsq < cutforcesq) {
                    MD_FLOAT sr2   = (MD_FLOAT)(1.0) / rsq;
                    MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
                    MD_FLOAT force = (MD_FLOAT)(48.0) * sr6 * (sr6 - (MD_FLOAT)(0.5)) * sr2 *
                                 epsilon;
                    MD_FLOAT fx = delx * force;
                    MD_FLOAT fy = dely * force;
                    MD_FLOAT fz = delz * force;

                    fx_acc[sci_ci] += fx;
                    fy_acc[sci_ci] += fy;
                    fz_acc[sci_ci] += fz;

                    if (half_neigh) {
                        atomicAdd(&cj_f[CL_X_OFFSET + cjj], -fx);
                        atomicAdd(&cj_f[CL_Y_OFFSET + cjj], -fy);
                        atomicAdd(&cj_f[CL_Z_OFFSET + cjj], -fz);
                    }
                }
            }
        }
    }

    #pragma unroll
    for(int sci_ci = 0; sci_ci < SCLUSTER_SIZE; sci_ci++) {
        int ai = sci_ci * CLUSTER_M + cii;
        
        // If M is less than the warp size, we perform forces reduction via
        // warp shuffles instead of using atomics since it should be cheaper
        // It is very unlikely that M > 32, but we keep this check here to
        // avoid any issues in such situations
        #if false && CLUSTER_M <= 32
        MD_FLOAT fix  = fx_acc[sci_ci];
        MD_FLOAT fiy  = fy_acc[sci_ci];
        MD_FLOAT fiz  = fz_acc[sci_ci];
        unsigned mask = 0xffffffff;
        
        for (int offset = CLUSTER_N / 2; offset > 0; offset /= 2) {
            #ifdef CUDA_TARGET
            fix += __shfl_down_sync(mask, fix, offset);
            fiy += __shfl_down_sync(mask, fiy, offset);
            fiz += __shfl_down_sync(mask, fiz, offset);
            #else
            fix += __shfl_down(fix, offset);
            fiy += __shfl_down(fiy, offset);
            fiz += __shfl_down(fiz, offset);
            #endif
        }

        if (cii == 0) {
            sci_f[CL_X_OFFSET + ai] += fix;
            sci_f[CL_Y_OFFSET + ai] += fiy;
            sci_f[CL_Z_OFFSET + ai] += fiz;
        }
        #else
        atomicAdd(&sci_f[CL_X_OFFSET + ai], fx_acc[sci_ci]);
        atomicAdd(&sci_f[CL_Y_OFFSET + ai], fy_acc[sci_ci]);
        atomicAdd(&sci_f[CL_Z_OFFSET + ai], fz_acc[sci_ci]);
        #endif
    }
}

__global__ void cudaUpdatePbcSup_warp(MD_FLOAT* cuda_cl_x,
    int* cuda_border_map,
    int* cuda_jclusters_natoms,
    int* cuda_PBCx,
    int* cuda_PBCy,
    int* cuda_PBCz,
    int Nclusters_local,
    int Nclusters_ghost,
    MD_FLOAT param_xprd,
    MD_FLOAT param_yprd,
    MD_FLOAT param_zprd) {

    int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) {
        return;
    }

    int ncj             = Nclusters_local * SCLUSTER_SIZE;
    int cj              = ncj + cg;
    int cj_vec_base     = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base   = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT* cj_x      = &cuda_cl_x[cj_vec_base];
    MD_FLOAT* bmap_x    = &cuda_cl_x[bmap_vec_base];

    for (int cjj = 0; cjj < CLUSTER_N; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * param_xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * param_yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * param_zprd;
    }
}

extern "C" double computeForceLJCudaSup(Parameter* param, Atom* atom, Neighbor* neighbor, Stats* stats) {
    DEBUG_MESSAGE("computeForceLJCudaSup start\r\n");

    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6     = param->sigma6;
    MD_FLOAT epsilon    = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * SCLUSTER_SIZE * 3 * sizeof(MD_FLOAT));
    dim3 block_size       = dim3(CLUSTER_M, CLUSTER_N, 1);
    dim3 grid_size        = dim3(atom->Nclusters_local, 1, 1);
    double S              = getTimeStamp();
    LIKWID_MARKER_START("force");

    computeForceLJCudaSup_warp<<<grid_size, block_size>>>(cuda_cl_x,
        cuda_cl_f,
        atom->Nclusters_local,
        cuda_numneigh,
        cuda_neighbors,
        neighbor->half_neigh,
        neighbor->maxneighs,
        cutforcesq,
        sigma6,
        epsilon);

    cuda_assert("computeForceLJCudaSup", hipPeekAtLastError());
    cuda_assert("computeForceLJCudaSup", hipDeviceSynchronize());

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    DEBUG_MESSAGE("computeForceLJCudaSup stop\r\n");
    return E - S;
}
