#include "hip/hip_runtime.h"
extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <force.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>
}

extern "C" {
extern MD_FLOAT* cuda_cl_x;
extern MD_FLOAT* cuda_cl_v;
extern MD_FLOAT* cuda_cl_f;
extern int* cuda_neighbors;
extern int* cuda_numneigh;
extern int* cuda_natoms;
extern int* natoms;
extern int* ngatoms;
extern int* cuda_border_map;
extern int* cuda_jclusters_natoms;
extern MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
extern MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
extern MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
extern int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
extern int* cuda_nclusters;
}

__global__ void cudaInitialIntegrateSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int* cuda_nclusters,
    int* cuda_natoms,
    int Nclusters_local,
    MD_FLOAT dtforce,
    MD_FLOAT dt) {

    int sci = blockDim.x * blockIdx.x + threadIdx.x;
    int ci = threadIdx.y;
    int cii = threadIdx.z;

    if (sci >= Nclusters_local) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    int i            = ci * CLUSTER_M + cii;
    MD_FLOAT* ci_x   = &cuda_cl_x[sci_vec_base];
    MD_FLOAT* ci_v   = &cuda_cl_v[sci_vec_base];
    MD_FLOAT* ci_f   = &cuda_cl_f[sci_vec_base];

    ci_v[SCL_X_OFFSET + i] += dtforce * ci_f[SCL_X_OFFSET + i];
    ci_v[SCL_Y_OFFSET + i] += dtforce * ci_f[SCL_Y_OFFSET + i];
    ci_v[SCL_Z_OFFSET + i] += dtforce * ci_f[SCL_Z_OFFSET + i];
    ci_x[SCL_X_OFFSET + i] += dt * ci_v[SCL_X_OFFSET + i];
    ci_x[SCL_Y_OFFSET + i] += dt * ci_v[SCL_Y_OFFSET + i];
    ci_x[SCL_Z_OFFSET + i] += dt * ci_v[SCL_Z_OFFSET + i];
}

__global__ void cudaFinalIntegrateSup_warp(MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int* cuda_nclusters,
    int* cuda_natoms,
    int Nclusters_local,
    MD_FLOAT dtforce) {

    int sci = blockDim.x * blockIdx.x + threadIdx.x;
    int ci = threadIdx.y;
    int cii = threadIdx.z;

    if (sci >= Nclusters_local) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);
    int i            = ci * CLUSTER_M + cii;
    MD_FLOAT* ci_v   = &cuda_cl_v[sci_vec_base];
    MD_FLOAT* ci_f   = &cuda_cl_f[sci_vec_base];

    ci_v[SCL_X_OFFSET + i] += dtforce * ci_f[SCL_X_OFFSET + i];
    ci_v[SCL_Y_OFFSET + i] += dtforce * ci_f[SCL_Y_OFFSET + i];
    ci_v[SCL_Z_OFFSET + i] += dtforce * ci_f[SCL_Z_OFFSET + i];
}

__global__ void computeForceLJCudaSup_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_f,
    int* cuda_nclusters,
    int Nclusters_local,
    int* cuda_numneigh,
    int* cuda_neighs,
    int half_neigh,
    int maxneighs,
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon) {

    int sci = blockIdx.x;
    //int ci = sci * SCLUSTER_SIZE + threadIdx.y;
    int cii = threadIdx.x;
    int cjj = threadIdx.y;

    if ((sci >= Nclusters_local) || (cii >= CLUSTER_M) || (cjj >= CLUSTER_N)) {
        return;
    }

    int sci_vec_base = SCI_VECTOR_BASE_INDEX(sci);

    for (int k = 0; k < cuda_numneigh[sci]; k++) {
        int cj = cuda_neighs[sci * maxneighs + k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT* cj_x  = &cuda_cl_x[cj_vec_base];
        MD_FLOAT* cj_f  = &cuda_cl_f[cj_vec_base];
        MD_FLOAT xjtmp = cj_x[SCL_X_OFFSET + cjj];
        MD_FLOAT yjtmp = cj_x[SCL_Y_OFFSET + cjj];
        MD_FLOAT zjtmp = cj_x[SCL_Z_OFFSET + cjj];

        for(int ci = 0; ci < cuda_nclusters[sci]; ci++) {
            int cond = cj / SCLUSTER_SIZE != sci ||
                    threadIdx.y != cj % SCLUSTER_SIZE ||
                    threadIdx.x != cjj;

            if (cond) {
                MD_FLOAT* ci_x  = &cuda_cl_x[sci_vec_base + ci * CLUSTER_M];
                MD_FLOAT* ci_f  = &cuda_cl_f[sci_vec_base + ci * CLUSTER_M];
                MD_FLOAT delx   = ci_x[SCL_X_OFFSET + cii] - xjtmp;
                MD_FLOAT dely   = ci_x[SCL_Y_OFFSET + cii] - yjtmp;
                MD_FLOAT delz   = ci_x[SCL_Z_OFFSET + cii] - zjtmp;
                MD_FLOAT rsq    = delx * delx + dely * dely + delz * delz;

                if (rsq < cutforcesq) {
                    MD_FLOAT sr2   = (MD_FLOAT)1.0 / rsq;
                    MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
                    MD_FLOAT force = (MD_FLOAT)48.0 * sr6 * (sr6 - (MD_FLOAT)0.5) * sr2 * epsilon;

                    if (half_neigh) {
                        atomicAdd(&cj_f[SCL_X_OFFSET + cjj], -delx * force);
                        atomicAdd(&cj_f[SCL_Y_OFFSET + cjj], -dely * force);
                        atomicAdd(&cj_f[SCL_Z_OFFSET + cjj], -delz * force);
                    }

                    atomicAdd(&ci_f[SCL_X_OFFSET + cii], delx * force);
                    atomicAdd(&ci_f[SCL_Y_OFFSET + cii], dely * force);
                    atomicAdd(&ci_f[SCL_Z_OFFSET + cii], delz * force);
                }
            }
        }
    }
}

__global__ void cudaUpdatePbcSup_warp(MD_FLOAT* cuda_cl_x,
    int* cuda_border_map,
    int* cuda_jclusters_natoms,
    int* cuda_PBCx,
    int* cuda_PBCy,
    int* cuda_PBCz,
    int Nclusters_local,
    int Nclusters_ghost,
    MD_FLOAT param_xprd,
    MD_FLOAT param_yprd,
    MD_FLOAT param_zprd)
{
    int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) return;

    // int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int jfac      = SCLUSTER_SIZE / CLUSTER_M;
    int ncj       = Nclusters_local / jfac;
    MD_FLOAT xprd = param_xprd;
    MD_FLOAT yprd = param_yprd;
    MD_FLOAT zprd = param_zprd;

    const int cj      = ncj + cg;
    int cj_vec_base   = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT* cj_x    = &cuda_cl_x[cj_vec_base];
    MD_FLOAT* bmap_x  = &cuda_cl_x[bmap_vec_base];

    for (int cjj = 0; cjj < cuda_jclusters_natoms[cg]; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * zprd;
    }
}

extern "C" double computeForceLJCudaSup(Parameter* param, Atom* atom, Neighbor* neighbor, Stats* stats) {
    DEBUG_MESSAGE("computeForceLJCudaSup start\r\n");

    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6     = param->sigma6;
    MD_FLOAT epsilon    = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * SCLUSTER_SIZE * 3 * sizeof(MD_FLOAT));
    dim3 block_size       = dim3(CLUSTER_M, CLUSTER_N, 1);
    dim3 grid_size        = dim3(atom->Nclusters_local, 1, 1);
    double S              = getTimeStamp();
    LIKWID_MARKER_START("force");

    computeForceLJCudaSup_warp<<<grid_size, block_size>>>(cuda_cl_x,
        cuda_cl_f,
        cuda_nclusters,
        atom->Nclusters_local,
        cuda_numneigh,
        cuda_neighbors,
        neighbor->half_neigh,
        neighbor->maxneighs,
        cutforcesq,
        sigma6,
        epsilon);

    cuda_assert("computeForceLJCudaSup", hipPeekAtLastError());
    cuda_assert("computeForceLJCudaSup", hipDeviceSynchronize());

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    DEBUG_MESSAGE("computeForceCudaSup stop\r\n");
    return E - S;
}
