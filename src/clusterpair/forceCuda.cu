#include "hip/hip_runtime.h"
/*
 * Copyright (C)  NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
extern "C" {
#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <force.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>
}

extern "C" {
MD_FLOAT* cuda_cl_x;
MD_FLOAT* cuda_cl_v;
MD_FLOAT* cuda_cl_f;
int* cuda_neighbors;
int* cuda_numneigh;
int* cuda_natoms;
int* natoms;
int* ngatoms;
int* cuda_border_map;
int* cuda_jclusters_natoms;
int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;

#ifndef ONE_ATOM_TYPE
int* cuda_cl_t;
MD_FLOAT* cuda_cutforcesq;
MD_FLOAT* cuda_sigma6;
MD_FLOAT* cuda_epsilon;
#endif
}

extern "C" void initDevice(Atom* atom, Neighbor* neighbor)
{
    cuda_assert("cudaDeviceSetup", hipDeviceReset());
    cuda_assert("cudaDeviceSetup", hipSetDevice(0));

    cuda_cl_x = (MD_FLOAT*)allocateGPU(
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_cl_v = (MD_FLOAT*)allocateGPU(
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_cl_f = (MD_FLOAT*)allocateGPU(
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
#ifndef ONE_ATOM_TYPE
    cuda_cl_t       = (int*)allocateGPU(atom->Nclusters_max * CLUSTER_M * sizeof(int));
    cuda_cutforcesq = (MD_FLOAT*)allocateGPU(
        atom->ntypes * atom->ntypes * sizeof(MD_FLOAT));
    cuda_sigma6  = (MD_FLOAT*)allocateGPU(atom->ntypes * atom->ntypes * sizeof(MD_FLOAT));
    cuda_epsilon = (MD_FLOAT*)allocateGPU(atom->ntypes * atom->ntypes * sizeof(MD_FLOAT));

    memcpyToGPU(cuda_cutforcesq,
        atom->cutforcesq,
        atom->ntypes * atom->ntypes * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_sigma6,
        atom->sigma6,
        atom->ntypes * atom->ntypes * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_epsilon,
        atom->epsilon,
        atom->ntypes * atom->ntypes * sizeof(MD_FLOAT));
#endif
    cuda_natoms           = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_jclusters_natoms = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_border_map       = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCx             = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCy             = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCz             = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_numneigh         = (int*)allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_neighbors        = (int*)allocateGPU(
        atom->Nclusters_max * neighbor->maxneighs * sizeof(int));
    natoms          = (int*)malloc(atom->Nclusters_max * sizeof(int));
    ngatoms         = (int*)malloc(atom->Nclusters_max * sizeof(int));
}

extern "C" void copyDataToCUDADevice(Atom* atom, Neighbor* neighbor)
{
    memcpyToGPU(cuda_cl_x,
        atom->cl_x,
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_cl_v,
        atom->cl_v,
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
#ifndef ONE_ATOM_TYPE
    memcpyToGPU(cuda_cl_t, atom->cl_t, atom->Nclusters_max * CLUSTER_M * sizeof(int));
#endif

    for (int ci = 0; ci < atom->Nclusters_local; ci++) {
        natoms[ci] = atom->iclusters[ci].natoms;
    }

    int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int ncj  = atom->Nclusters_local / jfac;
    for (int cg = 0; cg < atom->Nclusters_ghost; cg++) {
        const int cj = ncj + cg;
        ngatoms[cg]  = atom->jclusters[cj].natoms;
    }

    memcpyToGPU(cuda_natoms, natoms, atom->Nclusters_local * sizeof(int));
    memcpyToGPU(cuda_jclusters_natoms, ngatoms, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_border_map, atom->border_map, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCx, atom->PBCx, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCy, atom->PBCy, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCz, atom->PBCz, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_numneigh, neighbor->numneigh, atom->Nclusters_local * sizeof(int));
    memcpyToGPU(cuda_neighbors,
        neighbor->neighbors,
        atom->Nclusters_local * neighbor->maxneighs * sizeof(int));
}

extern "C" void copyDataFromCUDADevice(Atom* atom)
{
    memcpyFromGPU(atom->cl_x,
        cuda_cl_x,
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->cl_v,
        cuda_cl_v,
        atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
}

extern "C" void cudaDeviceFree(void)
{
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_x));
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_v));
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_f));
#ifndef ONE_ATOM_TYPE
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_t));
#endif
    cuda_assert("cudaDeviceFree", hipFree(cuda_numneigh));
    cuda_assert("cudaDeviceFree", hipFree(cuda_neighbors));
    cuda_assert("cudaDeviceFree", hipFree(cuda_natoms));
    cuda_assert("cudaDeviceFree", hipFree(cuda_border_map));
    cuda_assert("cudaDeviceFree", hipFree(cuda_jclusters_natoms));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCx));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCy));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCz));
    free(natoms);
    free(ngatoms);
}

__global__ void cudaInitialIntegrate_warp(MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int* cuda_natoms,
    int Nclusters_local,
    MD_FLOAT dtforce,
    MD_FLOAT dt)
{

    unsigned int ci = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci >= Nclusters_local) return;

    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci);
    MD_FLOAT* ci_x  = &cuda_cl_x[ci_vec_base];
    MD_FLOAT* ci_v  = &cuda_cl_v[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];

    for (int cii = 0; cii < cuda_natoms[ci]; cii++) {
        ci_v[CL_X_OFFSET + cii] += dtforce * ci_f[CL_X_OFFSET + cii];
        ci_v[CL_Y_OFFSET + cii] += dtforce * ci_f[CL_Y_OFFSET + cii];
        ci_v[CL_Z_OFFSET + cii] += dtforce * ci_f[CL_Z_OFFSET + cii];
        ci_x[CL_X_OFFSET + cii] += dt * ci_v[CL_X_OFFSET + cii];
        ci_x[CL_Y_OFFSET + cii] += dt * ci_v[CL_Y_OFFSET + cii];
        ci_x[CL_Z_OFFSET + cii] += dt * ci_v[CL_Z_OFFSET + cii];
    }
}

__global__ void cudaUpdatePbc_warp(MD_FLOAT* cuda_cl_x,
    int* cuda_border_map,
    int* cuda_jclusters_natoms,
    int* cuda_PBCx,
    int* cuda_PBCy,
    int* cuda_PBCz,
    int Nclusters_local,
    int Nclusters_ghost,
    MD_FLOAT param_xprd,
    MD_FLOAT param_yprd,
    MD_FLOAT param_zprd)
{
    unsigned int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) return;

    int jfac      = MAX(1, CLUSTER_N / CLUSTER_M);
    int ncj       = Nclusters_local / jfac;
    MD_FLOAT xprd = param_xprd;
    MD_FLOAT yprd = param_yprd;
    MD_FLOAT zprd = param_zprd;

    const int cj      = ncj + cg;
    int cj_vec_base   = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT* cj_x    = &cuda_cl_x[cj_vec_base];
    MD_FLOAT* bmap_x  = &cuda_cl_x[bmap_vec_base];

    for (int cjj = 0; cjj < cuda_jclusters_natoms[cg]; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * zprd;
    }
}

__global__ void computeForceLJCudaFullNeigh(
#ifdef ONE_ATOM_TYPE
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon,
#else
    int* cuda_cl_t,
    MD_FLOAT* atom_cutforcesq,
    MD_FLOAT* atom_sigma6,
    MD_FLOAT* atom_epsilon,
    int ntypes,
#endif
    MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    int Nclusters_max,
    int* cuda_numneigh,
    int* cuda_neighs,
    int maxneighs)
{

    int ci = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci >= Nclusters_local) return;

    int cii         = threadIdx.z;
    int cjj         = threadIdx.y;
    int ci_cj0      = CJ0_FROM_CI(ci);
    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci);
    MD_FLOAT* ci_x  = &cuda_cl_x[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];
    int* neighs     = &cuda_neighs[ci * maxneighs];
    int numneighs   = cuda_numneigh[ci];
    MD_FLOAT xtmp   = ci_x[CL_X_OFFSET + cii];
    MD_FLOAT ytmp   = ci_x[CL_Y_OFFSET + cii];
    MD_FLOAT ztmp   = ci_x[CL_Z_OFFSET + cii];
    MD_FLOAT fix    = 0;
    MD_FLOAT fiy    = 0;
    MD_FLOAT fiz    = 0;

#ifndef ONE_ATOM_TYPE
    int ci_sca_base = CI_SCALAR_BASE_INDEX(ci);
    int type_i      = cuda_cl_t[ci_sca_base + cii];
#endif

    for (int k = 0; k < numneighs; k++) {
        int cj          = neighs[k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT* cj_x  = &cuda_cl_x[cj_vec_base];

        int cond;
#if CLUSTER_M == CLUSTER_N
        cond = ci_cj0 != cj || cii != cjj;
#elif CLUSTER_M < CLUSTER_N
        cond = ci_cj0 != cj || cii + CLUSTER_M * (ci & 0x1) != cjj;
#endif
        if (cond) {
            MD_FLOAT delx = xtmp - cj_x[CL_X_OFFSET + cjj];
            MD_FLOAT dely = ytmp - cj_x[CL_Y_OFFSET + cjj];
            MD_FLOAT delz = ztmp - cj_x[CL_Z_OFFSET + cjj];
            MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

#ifndef ONE_ATOM_TYPE
            int cj_sca_base     = CJ_SCALAR_BASE_INDEX(cj);
            int type_j          = cuda_cl_t[cj_sca_base + cjj];
            int type_index      = type_i * ntypes + type_j;
            MD_FLOAT cutforcesq = atom_cutforcesq[type_index];
            MD_FLOAT sigma6     = atom_sigma6[type_index];
            MD_FLOAT epsilon    = atom_epsilon[type_index];
#endif

            if (rsq < cutforcesq) {
                MD_FLOAT sr2   = (MD_FLOAT)(1.0) / rsq;
                MD_FLOAT sr6   = sr2 * sr2 * sr2 * sigma6;
                MD_FLOAT force = (MD_FLOAT)(48.0) * sr6 * (sr6 - (MD_FLOAT)(0.5)) * sr2 * epsilon;

                fix += delx * force;
                fiy += dely * force;
                fiz += delz * force;
            }
        }
    }

// If M is less than the warp size, we perform forces reduction via
// warp shuffles instead of using atomics since it should be cheaper
// It is very unlikely that M > 32, but we keep this check here to
// avoid any issues in such situations
#if CLUSTER_M <= 32
    unsigned mask = 0xffffffff;
    for (int offset = CLUSTER_M / 2; offset > 0; offset /= 2) {
        fix += __shfl_down_sync(mask, fix, offset);
        fiy += __shfl_down_sync(mask, fiy, offset);
        fiz += __shfl_down_sync(mask, fiz, offset);
    }

    if (threadIdx.x == 0) {
        ci_f[CL_X_OFFSET + cii] = fix;
        ci_f[CL_Y_OFFSET + cii] = fiy;
        ci_f[CL_Z_OFFSET + cii] = fiz;
    }
#else
    atomicAdd(&ci_f[CL_X_OFFSET + cii], fix);
    atomicAdd(&ci_f[CL_Y_OFFSET + cii], fiy);
    atomicAdd(&ci_f[CL_Z_OFFSET + cii], fiz);
#endif
}

__global__ void computeForceLJCudaHalfNeigh(
#ifdef ONE_ATOM_TYPE
    MD_FLOAT cutforcesq,
    MD_FLOAT sigma6,
    MD_FLOAT epsilon,
#else
    int* cuda_cl_t,
    MD_FLOAT* atom_cutforcesq,
    MD_FLOAT* atom_sigma6,
    MD_FLOAT* atom_epsilon,
    int ntypes,
#endif
    MD_FLOAT* cuda_cl_x,
    MD_FLOAT* cuda_cl_f,
    int Nclusters_local,
    int Nclusters_max,
    int* cuda_numneigh,
    int* cuda_neighs,
    int maxneighs)
{
    int ci = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci >= Nclusters_local) return;

    int cii         = threadIdx.z;
    int cjj         = threadIdx.y;
    int ci_cj0      = CJ0_FROM_CI(ci);
    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci);
    MD_FLOAT* ci_x  = &cuda_cl_x[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];
    int* neighs     = &cuda_neighs[ci * maxneighs];
    int numneighs   = cuda_numneigh[ci];
    MD_FLOAT xtmp   = ci_x[CL_X_OFFSET + cii];
    MD_FLOAT ytmp   = ci_x[CL_Y_OFFSET + cii];
    MD_FLOAT ztmp   = ci_x[CL_Z_OFFSET + cii];
    MD_FLOAT fix    = 0;
    MD_FLOAT fiy    = 0;
    MD_FLOAT fiz    = 0;

#ifndef ONE_ATOM_TYPE
    int ci_sca_base = CI_SCALAR_BASE_INDEX(ci);
    int type_i      = cuda_cl_t[ci_sca_base + cii];
#endif

    for (int k = 0; k < numneighs; k++) {
        int cj          = neighs[k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT* cj_x  = &cuda_cl_x[cj_vec_base];
        MD_FLOAT* cj_f  = &cuda_cl_f[cj_vec_base];

        int cond;
#if CLUSTER_M == CLUSTER_N
        cond = ci_cj0 != cj || cii < cjj;
#elif CLUSTER_M < CLUSTER_N
        cond = ci_cj0 != cj || cii + CLUSTER_M * (ci & 0x1) < cjj;
#endif
        if (cond) {
            MD_FLOAT delx = xtmp - cj_x[CL_X_OFFSET + cjj];
            MD_FLOAT dely = ytmp - cj_x[CL_Y_OFFSET + cjj];
            MD_FLOAT delz = ztmp - cj_x[CL_Z_OFFSET + cjj];
            MD_FLOAT rsq  = delx * delx + dely * dely + delz * delz;

#ifndef ONE_ATOM_TYPE
            int cj_sca_base     = CJ_SCALAR_BASE_INDEX(cj);
            int type_j          = cuda_cl_t[cj_sca_base + cjj];
            int type_index      = type_i * ntypes + type_j;
            MD_FLOAT cutforcesq = atom_cutforcesq[type_index];
            MD_FLOAT sigma6     = atom_sigma6[type_index];
            MD_FLOAT epsilon    = atom_epsilon[type_index];
#endif

            if (rsq < cutforcesq) {
                MD_FLOAT sr2             = 1.0 / rsq;
                MD_FLOAT sr6             = sr2 * sr2 * sr2 * sigma6;
                MD_FLOAT force           = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;
                MD_FLOAT partial_force_x = delx * force;
                MD_FLOAT partial_force_y = dely * force;
                MD_FLOAT partial_force_z = delz * force;

                atomicAdd(&cj_f[CL_X_OFFSET + cjj], -partial_force_x);
                atomicAdd(&cj_f[CL_Y_OFFSET + cjj], -partial_force_y);
                atomicAdd(&cj_f[CL_Z_OFFSET + cjj], -partial_force_z);

                fix += partial_force_x;
                fiy += partial_force_y;
                fiz += partial_force_z;
            }
        }
    }

    atomicAdd(&ci_f[CL_X_OFFSET + cii], fix);
    atomicAdd(&ci_f[CL_Y_OFFSET + cii], fiy);
    atomicAdd(&ci_f[CL_Z_OFFSET + cii], fiz);
}

__global__ void cudaFinalIntegrate_warp(MD_FLOAT* cuda_cl_v,
    MD_FLOAT* cuda_cl_f,
    int* cuda_natoms,
    int Nclusters_local,
    MD_FLOAT dtforce)
{

    unsigned int ci = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci >= Nclusters_local) return;

    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci);
    MD_FLOAT* ci_v  = &cuda_cl_v[ci_vec_base];
    MD_FLOAT* ci_f  = &cuda_cl_f[ci_vec_base];

    for (int cii = 0; cii < cuda_natoms[ci]; cii++) {
        ci_v[CL_X_OFFSET + cii] += dtforce * ci_f[CL_X_OFFSET + cii];
        ci_v[CL_Y_OFFSET + cii] += dtforce * ci_f[CL_Y_OFFSET + cii];
        ci_v[CL_Z_OFFSET + cii] += dtforce * ci_f[CL_Z_OFFSET + cii];
    }
}

extern "C" void initialIntegrateCUDA(Parameter* param, Atom* atom)
{
    const int threads_num = 64;
    dim3 block_size       = dim3(threads_num, 1, 1);
    dim3 grid_size = dim3((atom->Nclusters_local + threads_num - 1) / threads_num, 1, 1);

    cudaInitialIntegrate_warp<<<grid_size, block_size>>>(cuda_cl_x,
        cuda_cl_v,
        cuda_cl_f,
        cuda_natoms,
        atom->Nclusters_local,
        param->dtforce,
        param->dt);

    cuda_assert("cudaInitialIntegrate", hipPeekAtLastError());
    cuda_assert("cudaInitialIntegrate", hipDeviceSynchronize());
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
extern "C" void updatePbcCUDA(Atom* atom, Parameter* param)
{
    const int threads_num = 64;
    dim3 block_size       = dim3(threads_num, 1, 1);
    dim3 grid_size = dim3((atom->Nclusters_ghost + threads_num - 1) / threads_num, 1, 1);

    cudaUpdatePbc_warp<<<grid_size, block_size>>>(cuda_cl_x,
        cuda_border_map,
        cuda_jclusters_natoms,
        cuda_PBCx,
        cuda_PBCy,
        cuda_PBCz,
        atom->Nclusters_local,
        atom->Nclusters_ghost,
        param->xprd,
        param->yprd,
        param->zprd);

    cuda_assert("cudaUpdatePbc", hipPeekAtLastError());
    cuda_assert("cudaUpdatePbc", hipDeviceSynchronize());
}

extern "C" double computeForceLJCUDA(
    Parameter* param, Atom* atom, Neighbor* neighbor, Stats* stats)
{
#ifdef ONE_ATOM_TYPE
    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6     = param->sigma6;
    MD_FLOAT epsilon    = param->epsilon;
#endif

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_local * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    const int threads_num = 1;
    dim3 block_size       = dim3(threads_num, CLUSTER_N, CLUSTER_M);
    dim3 grid_size = dim3((atom->Nclusters_local + threads_num - 1) / threads_num, 1, 1);
    double S       = getTimeStamp();
    LIKWID_MARKER_START("force");

    if (neighbor->half_neigh) {
        computeForceLJCudaHalfNeigh<<<grid_size, block_size>>>(
#ifdef ONE_ATOM_TYPE
            cutforcesq,
            sigma6,
            epsilon,
#else
            cuda_cl_t,
            cuda_cutforcesq,
            cuda_sigma6,
            cuda_epsilon,
            atom->ntypes,
#endif
            cuda_cl_x,
            cuda_cl_f,
            atom->Nclusters_local,
            atom->Nclusters_max,
            cuda_numneigh,
            cuda_neighbors,
            neighbor->maxneighs);
    } else {
        computeForceLJCudaFullNeigh<<<grid_size, block_size>>>(
#ifdef ONE_ATOM_TYPE
            cutforcesq,
            sigma6,
            epsilon,
#else
            cuda_cl_t,
            cuda_cutforcesq,
            cuda_sigma6,
            cuda_epsilon,
            atom->ntypes,
#endif
            cuda_cl_x,
            cuda_cl_f,
            atom->Nclusters_local,
            atom->Nclusters_max,
            cuda_numneigh,
            cuda_neighbors,
            neighbor->maxneighs);
    }

    cuda_assert("computeForceLJ_cuda", hipPeekAtLastError());
    cuda_assert("computeForceLJ_cuda", hipDeviceSynchronize());
    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    return E - S;
}

extern "C" void finalIntegrateCUDA(Parameter* param, Atom* atom)
{
    const int threads_num = 64;
    dim3 block_size       = dim3(threads_num, 1, 1);
    dim3 grid_size = dim3((atom->Nclusters_local + threads_num - 1) / threads_num, 1, 1);

    cudaFinalIntegrate_warp<<<grid_size, block_size>>>(cuda_cl_v,
        cuda_cl_f,
        cuda_natoms,
        atom->Nclusters_local,
        param->dt);

    cuda_assert("cudaFinalIntegrate", hipPeekAtLastError());
    cuda_assert("cudaFinalIntegrate", hipDeviceSynchronize());
}
