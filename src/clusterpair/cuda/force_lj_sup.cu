#include "hip/hip_runtime.h"

extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>

}

extern "C" {
    extern MD_FLOAT *cuda_cl_x;
    extern MD_FLOAT *cuda_cl_v;
    extern MD_FLOAT *cuda_cl_f;
    extern int *cuda_neighbors;
    extern int *cuda_numneigh;
    extern int *cuda_natoms;
    extern int *natoms;
    extern int *ngatoms;
    extern int *cuda_border_map;
    extern int *cuda_jclusters_natoms;
    extern MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
    extern MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
    extern MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
    extern int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
    extern int isReneighboured;

    extern int *cuda_iclusters;
    extern int *cuda_nclusters;

    extern MD_FLOAT *cuda_scl_x;
    extern MD_FLOAT *cuda_scl_v;
    extern MD_FLOAT *cuda_scl_f;
}

#ifdef USE_SUPER_CLUSTERS
extern "C"
void alignDataToSuperclusters(Atom *atom) {
    for (int sci = 0; sci < atom->Nsclusters_local; sci++) {
        const unsigned int scl_offset = sci * SCLUSTER_SIZE * 3 * CLUSTER_M;

        for (int ci = 0, scci = scl_offset; ci < atom->siclusters[sci].nclusters; ci++, scci += CLUSTER_M) {

            MD_FLOAT *ci_x = &atom->cl_x[CI_VECTOR_BASE_INDEX(atom->icluster_idx[SCLUSTER_SIZE * sci + ci])];
            MD_FLOAT *ci_v = &atom->cl_v[CI_VECTOR_BASE_INDEX(atom->icluster_idx[SCLUSTER_SIZE * sci + ci])];
            MD_FLOAT *ci_f = &atom->cl_f[CI_VECTOR_BASE_INDEX(atom->icluster_idx[SCLUSTER_SIZE * sci + ci])];

            /*
            MD_FLOAT *ci_x = &atom->cl_x[CI_VECTOR_BASE_INDEX(atom->siclusters[sci].iclusters[ci])];
            MD_FLOAT *ci_v = &atom->cl_v[CI_VECTOR_BASE_INDEX(atom->siclusters[sci].iclusters[ci])];
            MD_FLOAT *ci_f = &atom->cl_f[CI_VECTOR_BASE_INDEX(atom->siclusters[sci].iclusters[ci])];
             */

            memcpy(&atom->scl_x[scci], &ci_x[0], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&atom->scl_x[scci + SCLUSTER_SIZE * CLUSTER_M], &ci_x[0 + CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&atom->scl_x[scci + 2 * SCLUSTER_SIZE * CLUSTER_M], &ci_x[0 + 2 * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));

            memcpy(&atom->scl_v[scci], &ci_v[0], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&atom->scl_v[scci + SCLUSTER_SIZE * CLUSTER_M], &ci_v[0 + CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&atom->scl_v[scci + 2 * SCLUSTER_SIZE * CLUSTER_M], &ci_v[0 + 2 * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));

            memcpy(&atom->scl_f[scci], &ci_f[0], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&atom->scl_f[scci + SCLUSTER_SIZE * CLUSTER_M], &ci_f[0 + CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&atom->scl_f[scci + 2 * SCLUSTER_SIZE * CLUSTER_M], &ci_f[0 + 2 * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));

        }
    }
}

extern "C"
void alignDataFromSuperclusters(Atom *atom) {
    for (int sci = 0; sci < atom->Nsclusters_local; sci++) {
        const unsigned int scl_offset = sci * SCLUSTER_SIZE * 3 * CLUSTER_M;

        for (int ci = 0, scci = scl_offset; ci < atom->siclusters[sci].nclusters; ci++, scci += CLUSTER_M) {


            MD_FLOAT *ci_x = &atom->cl_x[CI_VECTOR_BASE_INDEX(atom->icluster_idx[SCLUSTER_SIZE * sci + ci])];
            MD_FLOAT *ci_v = &atom->cl_v[CI_VECTOR_BASE_INDEX(atom->icluster_idx[SCLUSTER_SIZE * sci + ci])];
            MD_FLOAT *ci_f = &atom->cl_f[CI_VECTOR_BASE_INDEX(atom->icluster_idx[SCLUSTER_SIZE * sci + ci])];

            /*
            MD_FLOAT *ci_x = &atom->cl_x[CI_VECTOR_BASE_INDEX(atom->siclusters[sci].iclusters[ci])];
            MD_FLOAT *ci_v = &atom->cl_v[CI_VECTOR_BASE_INDEX(atom->siclusters[sci].iclusters[ci])];
            MD_FLOAT *ci_f = &atom->cl_f[CI_VECTOR_BASE_INDEX(atom->siclusters[sci].iclusters[ci])];
             */

            memcpy(&ci_x[0], &atom->scl_x[scci], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&ci_x[0 + CLUSTER_M], &atom->scl_x[scci + SCLUSTER_SIZE * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&ci_x[0 + 2 * CLUSTER_M], &atom->scl_x[scci + 2 * SCLUSTER_SIZE * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));

            memcpy(&ci_v[0], &atom->scl_v[scci], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&ci_v[0 + CLUSTER_M], &atom->scl_v[scci + SCLUSTER_SIZE * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&ci_v[0 + 2 * CLUSTER_M], &atom->scl_v[scci + 2 * SCLUSTER_SIZE * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));

            memcpy(&ci_f[0], &atom->scl_f[scci], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&ci_f[0 + CLUSTER_M], &atom->scl_f[scci + SCLUSTER_SIZE * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));
            memcpy(&ci_f[0 + 2 * CLUSTER_M], &atom->scl_f[scci + 2 * SCLUSTER_SIZE * CLUSTER_M], CLUSTER_M * sizeof(MD_FLOAT));

        }
    }
}

__global__ void cudaInitialIntegrateSup_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                             int *cuda_nclusters,
                                             int *cuda_natoms,
                                             int Nsclusters_local, MD_FLOAT dtforce, MD_FLOAT dt) {

    unsigned int sci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    //unsigned int cii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    if (sci_pos >= Nsclusters_local) return;

    //unsigned int ci_pos = cii_pos / CLUSTER_M;
    //unsigned int scii_pos = cii_pos % CLUSTER_M;

    //if (ci_pos >= cuda_nclusters[sci_pos]) return;
    //if (scii_pos >= cuda_natoms[ci_pos]) return;

    int ci_vec_base = SCI_VECTOR_BASE_INDEX(sci_pos);
    MD_FLOAT *ci_x = &cuda_cl_x[ci_vec_base];
    MD_FLOAT *ci_v = &cuda_cl_v[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];

    for (int scii_pos = 0; scii_pos < SCLUSTER_M; scii_pos++) {
        ci_v[SCL_X_OFFSET + scii_pos] += dtforce * ci_f[SCL_X_OFFSET + scii_pos];
        ci_v[SCL_Y_OFFSET + scii_pos] += dtforce * ci_f[SCL_Y_OFFSET + scii_pos];
        ci_v[SCL_Z_OFFSET + scii_pos] += dtforce * ci_f[SCL_Z_OFFSET + scii_pos];
        ci_x[SCL_X_OFFSET + scii_pos] += dt * ci_v[SCL_X_OFFSET + scii_pos];
        ci_x[SCL_Y_OFFSET + scii_pos] += dt * ci_v[SCL_Y_OFFSET + scii_pos];
        ci_x[SCL_Z_OFFSET + scii_pos] += dt * ci_v[SCL_Z_OFFSET + scii_pos];
    }
}

__global__ void cudaFinalIntegrateSup_warp(MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                           int *cuda_nclusters, int *cuda_natoms,
                                           int Nsclusters_local, MD_FLOAT dtforce) {

    unsigned int sci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    //unsigned int cii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    if (sci_pos >= Nsclusters_local) return;

    //unsigned int ci_pos = cii_pos / CLUSTER_M;
    //unsigned int scii_pos = cii_pos % CLUSTER_M;

    //if (ci_pos >= cuda_nclusters[sci_pos]) return;
    //if (scii_pos >= cuda_natoms[ci_pos]) return;

    int ci_vec_base = SCI_VECTOR_BASE_INDEX(sci_pos);
    MD_FLOAT *ci_v = &cuda_cl_v[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];

    for (int scii_pos = 0; scii_pos < SCLUSTER_M; scii_pos++) {
        ci_v[SCL_X_OFFSET + scii_pos] += dtforce * ci_f[SCL_X_OFFSET + scii_pos];
        ci_v[SCL_Y_OFFSET + scii_pos] += dtforce * ci_f[SCL_Y_OFFSET + scii_pos];
        ci_v[SCL_Z_OFFSET + scii_pos] += dtforce * ci_f[SCL_Z_OFFSET + scii_pos];
    }

}

__global__ void computeForceLJSup_cuda_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_f,
                                            int *cuda_nclusters, int *cuda_iclusters,
                                            int Nsclusters_local,
                                            int *cuda_numneigh, int *cuda_neighs, int half_neigh, int maxneighs,
                                            MD_FLOAT cutforcesq, MD_FLOAT sigma6, MD_FLOAT epsilon) {

    unsigned int sci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int scii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int cjj_pos = blockDim.z * blockIdx.z + threadIdx.z;
    if ((sci_pos >= Nsclusters_local) || (scii_pos >= SCLUSTER_M) || (cjj_pos >= CLUSTER_N)) return;

    unsigned int ci_pos = scii_pos / CLUSTER_M;
    unsigned int cii_pos = scii_pos % CLUSTER_M;

    if (ci_pos >= cuda_nclusters[sci_pos]) return;

    int ci_cj0 = CJ0_FROM_CI(ci_pos);
    int ci_vec_base = SCI_VECTOR_BASE_INDEX(sci_pos);
    MD_FLOAT *ci_x = &cuda_cl_x[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];


    //int numneighs = cuda_numneigh[ci_pos];
    int numneighs = cuda_numneigh[cuda_iclusters[SCLUSTER_SIZE * sci_pos + ci_pos]];

    for(int k = 0; k < numneighs; k++) {
        int glob_j = (&cuda_neighs[cuda_iclusters[SCLUSTER_SIZE * sci_pos + ci_pos] * maxneighs])[k];
        int scj = glob_j / SCLUSTER_SIZE;
        // TODO Make cj accessible from super cluster data alignment (not reachable right now)
        int cj = SCJ_VECTOR_BASE_INDEX(scj) + CLUSTER_M * (glob_j % SCLUSTER_SIZE);
        int cj_vec_base = cj;
        MD_FLOAT *cj_x = &cuda_cl_x[cj_vec_base];
        MD_FLOAT *cj_f = &cuda_cl_f[cj_vec_base];

        MD_FLOAT xtmp = ci_x[SCL_CL_X_OFFSET(ci_pos) + cii_pos];
        MD_FLOAT ytmp = ci_x[SCL_CL_Y_OFFSET(ci_pos) + cii_pos];
        MD_FLOAT ztmp = ci_x[SCL_CL_Z_OFFSET(ci_pos) + cii_pos];
        MD_FLOAT fix = 0;
        MD_FLOAT fiy = 0;
        MD_FLOAT fiz = 0;


        //int cond = ci_cj0 != cj || cii_pos != cjj_pos || scj != sci_pos;
        int cond = (glob_j != cuda_iclusters[SCLUSTER_SIZE * sci_pos + ci_pos] && cii_pos != cjj_pos);

        if(cond) {
            MD_FLOAT delx = xtmp - cj_x[SCL_CL_X_OFFSET(ci_pos) + cjj_pos];
            MD_FLOAT dely = ytmp - cj_x[SCL_CL_Y_OFFSET(ci_pos) + cjj_pos];
            MD_FLOAT delz = ztmp - cj_x[SCL_CL_Z_OFFSET(ci_pos) + cjj_pos];
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;
            if(rsq < cutforcesq) {
                MD_FLOAT sr2 = 1.0 / rsq;
                MD_FLOAT sr6 = sr2 * sr2 * sr2 * sigma6;
                MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;

                if(half_neigh) {
                    atomicAdd(&cj_f[SCL_CL_X_OFFSET(ci_pos) + cjj_pos], -delx * force);
                    atomicAdd(&cj_f[SCL_CL_Y_OFFSET(ci_pos) + cjj_pos], -dely * force);
                    atomicAdd(&cj_f[SCL_CL_Z_OFFSET(ci_pos) + cjj_pos], -delz * force);
                }

                fix += delx * force;
                fiy += dely * force;
                fiz += delz * force;

                atomicAdd(&ci_f[SCL_CL_X_OFFSET(ci_pos) + cii_pos], fix);
                atomicAdd(&ci_f[SCL_CL_Y_OFFSET(ci_pos) + cii_pos], fiy);
                atomicAdd(&ci_f[SCL_CL_Z_OFFSET(ci_pos) + cii_pos], fiz);
            }
        }
    }

}

extern "C"
double computeForceLJSup_cuda(Parameter *param, Atom *atom, Neighbor *neighbor, Stats *stats) {
    DEBUG_MESSAGE("computeForceLJSup_cuda start\r\n");

    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6 = param->sigma6;
    MD_FLOAT epsilon = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    if (isReneighboured) {

        for(int ci = 0; ci < atom->Nclusters_local; ci++) {
            memcpyToGPU(&cuda_numneigh[ci], &neighbor->numneigh[ci], sizeof(int));
            memcpyToGPU(&cuda_neighbors[ci * neighbor->maxneighs], &neighbor->neighbors[ci * neighbor->maxneighs], neighbor->numneigh[ci] * sizeof(int));
        }

        for(int sci = 0; sci < atom->Nsclusters_local; sci++) {
            memcpyToGPU(&cuda_nclusters[sci], &atom->siclusters[sci].nclusters, sizeof(int));
            //memcpyToGPU(&cuda_iclusters[sci * SCLUSTER_SIZE], &atom->siclusters[sci].iclusters, sizeof(int) * atom->siclusters[sci].nclusters);
        }

        memcpyToGPU(cuda_iclusters, atom->icluster_idx, atom->Nsclusters_max * SCLUSTER_SIZE * sizeof(int));

        isReneighboured = 0;
    }

    const int threads_num = 1;
    dim3 block_size = dim3(threads_num, SCLUSTER_M, CLUSTER_N);
    dim3 grid_size = dim3(atom->Nsclusters_local/threads_num+1, 1, 1);
    double S = getTimeStamp();
    LIKWID_MARKER_START("force");
    computeForceLJSup_cuda_warp<<<grid_size, block_size>>>(cuda_scl_x, cuda_scl_f,
                                                           cuda_nclusters, cuda_iclusters,
                                                           atom->Nsclusters_local,
                                                           cuda_numneigh, cuda_neighbors,
                                                           neighbor->half_neigh, neighbor->maxneighs, cutforcesq,
                                                           sigma6, epsilon);
    cuda_assert("computeForceLJ_cuda", hipPeekAtLastError());
    cuda_assert("computeForceLJ_cuda", hipDeviceSynchronize());
    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    DEBUG_MESSAGE("computeForceLJSup_cuda stop\r\n");
    return E-S;
}
#endif //USE_SUPER_CLUSTERS
