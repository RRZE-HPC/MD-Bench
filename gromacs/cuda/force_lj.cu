#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>

}

extern "C" {
    MD_FLOAT *cuda_cl_x;
    MD_FLOAT *cuda_cl_v;
    MD_FLOAT *cuda_cl_f;
    int *cuda_neighbors;
    int *cuda_numneigh;
    int *cuda_natoms;
    int *natoms;
    int *ngatoms;
    int *cuda_border_map;
    int *cuda_jclusters_natoms;
    MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
    MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
    MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
    int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
    int isReneighboured;

    int *cuda_iclusters;
    int *cuda_nclusters;

    int cuda_max_scl;
    MD_FLOAT *cuda_scl_x;
    MD_FLOAT *cuda_scl_v;
    MD_FLOAT *cuda_scl_f;

    extern void alignDataToSuperclusters(Atom *atom);
    extern void alignDataFromSuperclusters(Atom *atom);
    extern double computeForceLJSup_cuda(Parameter *param, Atom *atom, Neighbor *neighbor, Stats *stats);
}

extern __global__ void cudaInitialIntegrateSup_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                                    int *cuda_nclusters,
                                                    int *cuda_natoms,
                                                    int Nsclusters_local, MD_FLOAT dtforce, MD_FLOAT dt);

extern __global__ void cudaFinalIntegrateSup_warp(MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                                  int *cuda_nclusters, int *cuda_natoms,
                                                  int Nsclusters_local, MD_FLOAT dtforce);

extern "C"
void initDevice(Atom *atom, Neighbor *neighbor) {
    cuda_assert("cudaDeviceSetup", hipDeviceReset());
    cuda_assert("cudaDeviceSetup", hipSetDevice(0));
    cuda_cl_x               =   (MD_FLOAT *) allocateGPU(atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_cl_v               =   (MD_FLOAT *) allocateGPU(atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_cl_f               =   (MD_FLOAT *) allocateGPU(atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_natoms             =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_jclusters_natoms   =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_border_map         =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCx               =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCy               =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCz               =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_numneigh           =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_neighbors          =   (int *) allocateGPU(atom->Nclusters_max * neighbor->maxneighs * sizeof(int));
    natoms                  =   (int *) malloc(atom->Nclusters_max * sizeof(int));
    ngatoms                 =   (int *) malloc(atom->Nclusters_max * sizeof(int));
    isReneighboured = 1;

#ifdef USE_SUPER_CLUSTERS
    cuda_max_scl            =   atom->Nsclusters_max;
    cuda_iclusters          =   (int *) allocateGPU(atom->Nsclusters_max * SCLUSTER_SIZE * sizeof(int));
    cuda_nclusters          =   (int *) allocateGPU(atom->Nsclusters_max * sizeof(int));

    cuda_scl_x              =   (MD_FLOAT *) allocateGPU(atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_scl_v              =   (MD_FLOAT *) allocateGPU(atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_scl_f              =   (MD_FLOAT *) allocateGPU(atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));

#endif //USE_SUPER_CLUSTERS
}

extern "C"
void copyDataToCUDADevice(Atom *atom) {
    DEBUG_MESSAGE("copyDataToCUDADevice start\r\n");

    memcpyToGPU(cuda_cl_x, atom->cl_x, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_cl_v, atom->cl_v, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_cl_f, atom->cl_f, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));

    for(int ci = 0; ci < atom->Nclusters_local; ci++) {
        natoms[ci] = atom->iclusters[ci].natoms;
    }

    memcpyToGPU(cuda_natoms, natoms, atom->Nclusters_local * sizeof(int));

    int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int ncj = atom->Nclusters_local / jfac;
    for(int cg = 0; cg < atom->Nclusters_ghost; cg++) {
        const int cj = ncj + cg;
        ngatoms[cg] = atom->jclusters[cj].natoms;
    }

    memcpyToGPU(cuda_jclusters_natoms, ngatoms, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_border_map, atom->border_map, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCx, atom->PBCx, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCy, atom->PBCy, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCz, atom->PBCz, atom->Nclusters_ghost * sizeof(int));

#ifdef USE_SUPER_CLUSTERS
    alignDataToSuperclusters(atom);

    if (cuda_max_scl < atom->Nsclusters_max) {
        cuda_assert("cudaDeviceFree", hipFree(cuda_scl_x));
        cuda_assert("cudaDeviceFree", hipFree(cuda_scl_v));
        cuda_assert("cudaDeviceFree", hipFree(cuda_scl_f));
        cuda_max_scl            =   atom->Nsclusters_max;

        cuda_iclusters          =   (int *) allocateGPU(atom->Nsclusters_max * SCLUSTER_SIZE * sizeof(int));
        cuda_nclusters          =   (int *) allocateGPU(atom->Nsclusters_max * sizeof(int));

        cuda_scl_x              =   (MD_FLOAT *) allocateGPU(atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
        cuda_scl_v              =   (MD_FLOAT *) allocateGPU(atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
        cuda_scl_f              =   (MD_FLOAT *) allocateGPU(atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    }
    memcpyToGPU(cuda_scl_x, atom->scl_x, atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_scl_v, atom->scl_v, atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_scl_f, atom->scl_f, atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
#endif //USE_SUPER_CLUSTERS

    DEBUG_MESSAGE("copyDataToCUDADevice stop\r\n");

}

extern "C"
void copyDataFromCUDADevice(Atom *atom) {
    DEBUG_MESSAGE("copyDataFromCUDADevice start\r\n");

    memcpyFromGPU(atom->cl_x, cuda_cl_x, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->cl_v, cuda_cl_v, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->cl_f, cuda_cl_f, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));

#ifdef USE_SUPER_CLUSTERS
    memcpyFromGPU(atom->scl_x, cuda_scl_x, atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->scl_v, cuda_scl_v, atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->scl_f, cuda_scl_f, atom->Nsclusters_max * SCLUSTER_M * 3 * sizeof(MD_FLOAT));

    alignDataFromSuperclusters(atom);
#endif //USE_SUPER_CLUSTERS

    DEBUG_MESSAGE("copyDataFromCUDADevice stop\r\n");
}

extern "C"
void cudaDeviceFree() {
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_x));
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_v));
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_f));
    cuda_assert("cudaDeviceFree", hipFree(cuda_numneigh));
    cuda_assert("cudaDeviceFree", hipFree(cuda_neighbors));
    cuda_assert("cudaDeviceFree", hipFree(cuda_natoms));
    cuda_assert("cudaDeviceFree", hipFree(cuda_border_map));
    cuda_assert("cudaDeviceFree", hipFree(cuda_jclusters_natoms));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCx));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCy));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCz));
    free(natoms);
    free(ngatoms);

#ifdef USE_SUPER_CLUSTERS
    cuda_assert("cudaDeviceFree", hipFree(cuda_scl_x));
    cuda_assert("cudaDeviceFree", hipFree(cuda_scl_v));
    cuda_assert("cudaDeviceFree", hipFree(cuda_scl_f));
#endif //USE_SUPER_CLUSTERS
}

__global__ void cudaInitialIntegrate_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                         int *cuda_natoms,
                                         int Nclusters_local, MD_FLOAT dtforce, MD_FLOAT dt) {

    unsigned int ci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci_pos >= Nclusters_local) return;

    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci_pos);
    MD_FLOAT *ci_x = &cuda_cl_x[ci_vec_base];
    MD_FLOAT *ci_v = &cuda_cl_v[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];

    for (int cii = 0; cii < cuda_natoms[ci_pos]; cii++) {
        ci_v[CL_X_OFFSET + cii] += dtforce * ci_f[CL_X_OFFSET + cii];
        ci_v[CL_Y_OFFSET + cii] += dtforce * ci_f[CL_Y_OFFSET + cii];
        ci_v[CL_Z_OFFSET + cii] += dtforce * ci_f[CL_Z_OFFSET + cii];
        ci_x[CL_X_OFFSET + cii] += dt * ci_v[CL_X_OFFSET + cii];
        ci_x[CL_Y_OFFSET + cii] += dt * ci_v[CL_Y_OFFSET + cii];
        ci_x[CL_Z_OFFSET + cii] += dt * ci_v[CL_Z_OFFSET + cii];
    }
}

__global__ void cudaUpdatePbc_warp(MD_FLOAT *cuda_cl_x, int *cuda_border_map,
                                   int *cuda_jclusters_natoms,
                                   int *cuda_PBCx,
                                   int *cuda_PBCy,
                                   int *cuda_PBCz,
                                   int Nclusters_local,
                                   int Nclusters_ghost,
                                   MD_FLOAT param_xprd,
                                   MD_FLOAT param_yprd,
                                   MD_FLOAT param_zprd) {
    unsigned int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) return;

    int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int ncj = Nclusters_local / jfac;
    MD_FLOAT xprd = param_xprd;
    MD_FLOAT yprd = param_yprd;
    MD_FLOAT zprd = param_zprd;

    const int cj = ncj + cg;
    int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT *cj_x = &cuda_cl_x[cj_vec_base];
    MD_FLOAT *bmap_x = &cuda_cl_x[bmap_vec_base];

    for(int cjj = 0; cjj < cuda_jclusters_natoms[cg]; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * zprd;
    }
}

__global__ void cudaUpdatePbcSup_warp(MD_FLOAT *cuda_cl_x, int *cuda_border_map,
                                   int *cuda_jclusters_natoms,
                                   int *cuda_PBCx,
                                   int *cuda_PBCy,
                                   int *cuda_PBCz,
                                   int Nsclusters_local,
                                   int Nclusters_ghost,
                                   MD_FLOAT param_xprd,
                                   MD_FLOAT param_yprd,
                                   MD_FLOAT param_zprd) {
    unsigned int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) return;

    //int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int jfac = SCLUSTER_SIZE / CLUSTER_M;
    int ncj = Nsclusters_local / jfac;
    MD_FLOAT xprd = param_xprd;
    MD_FLOAT yprd = param_yprd;
    MD_FLOAT zprd = param_zprd;

    const int cj = ncj + cg;
    int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT *cj_x = &cuda_cl_x[cj_vec_base];
    MD_FLOAT *bmap_x = &cuda_cl_x[bmap_vec_base];

    for(int cjj = 0; cjj < cuda_jclusters_natoms[cg]; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * zprd;
    }
}

__global__ void computeForceLJ_cuda_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_f,
                                         int Nclusters_local, int Nclusters_max,
                                         int *cuda_numneigh, int *cuda_neighs, int half_neigh, int maxneighs,
                                         MD_FLOAT cutforcesq, MD_FLOAT sigma6, MD_FLOAT epsilon) {

    unsigned int ci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int cii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int cjj_pos = blockDim.z * blockIdx.z + threadIdx.z;
    if ((ci_pos >= Nclusters_local) || (cii_pos >= CLUSTER_M) || (cjj_pos >= CLUSTER_N)) return;

    int ci_cj0 = CJ0_FROM_CI(ci_pos);
    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci_pos);
    MD_FLOAT *ci_x = &cuda_cl_x[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];
    int numneighs = cuda_numneigh[ci_pos];
    for(int k = 0; k < numneighs; k++) {
        int cj = (&cuda_neighs[ci_pos * maxneighs])[k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT *cj_x = &cuda_cl_x[cj_vec_base];
        MD_FLOAT *cj_f = &cuda_cl_f[cj_vec_base];

        MD_FLOAT xtmp = ci_x[CL_X_OFFSET + cii_pos];
        MD_FLOAT ytmp = ci_x[CL_Y_OFFSET + cii_pos];
        MD_FLOAT ztmp = ci_x[CL_Z_OFFSET + cii_pos];
        MD_FLOAT fix = 0;
        MD_FLOAT fiy = 0;
        MD_FLOAT fiz = 0;

        int cond;
#if CLUSTER_M == CLUSTER_N
        cond = half_neigh ? (ci_cj0 != cj || cii_pos < cjj_pos) :
                            (ci_cj0 != cj || cii_pos != cjj_pos);
#elif CLUSTER_M < CLUSTER_N
        cond = half_neigh ? (ci_cj0 != cj || cii_pos + CLUSTER_M * (ci_pos & 0x1) < cjj_pos) :
                            (ci_cj0 != cj || cii_pos + CLUSTER_M * (ci_pos & 0x1) != cjj_pos);
#endif
        if(cond) {
            MD_FLOAT delx = xtmp - cj_x[CL_X_OFFSET + cjj_pos];
            MD_FLOAT dely = ytmp - cj_x[CL_Y_OFFSET + cjj_pos];
            MD_FLOAT delz = ztmp - cj_x[CL_Z_OFFSET + cjj_pos];
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;
            if(rsq < cutforcesq) {
                MD_FLOAT sr2 = 1.0 / rsq;
                MD_FLOAT sr6 = sr2 * sr2 * sr2 * sigma6;
                MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;

                if(half_neigh) {
                    atomicAdd(&cj_f[CL_X_OFFSET + cjj_pos], -delx * force);
                    atomicAdd(&cj_f[CL_Y_OFFSET + cjj_pos], -dely * force);
                    atomicAdd(&cj_f[CL_Z_OFFSET + cjj_pos], -delz * force);
                }

                fix += delx * force;
                fiy += dely * force;
                fiz += delz * force;

                atomicAdd(&ci_f[CL_X_OFFSET + cii_pos], fix);
                atomicAdd(&ci_f[CL_Y_OFFSET + cii_pos], fiy);
                atomicAdd(&ci_f[CL_Z_OFFSET + cii_pos], fiz);
            }
        }
    }
}

__global__ void cudaFinalIntegrate_warp(MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                          int *cuda_natoms,
                                          int Nclusters_local, MD_FLOAT dtforce) {

    unsigned int ci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci_pos >= Nclusters_local) return;

    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci_pos);
    MD_FLOAT *ci_v = &cuda_cl_v[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];

    for (int cii = 0; cii < cuda_natoms[ci_pos]; cii++) {
        ci_v[CL_X_OFFSET + cii] += dtforce * ci_f[CL_X_OFFSET + cii];
        ci_v[CL_Y_OFFSET + cii] += dtforce * ci_f[CL_Y_OFFSET + cii];
        ci_v[CL_Z_OFFSET + cii] += dtforce * ci_f[CL_Z_OFFSET + cii];
    }
}

extern "C"
void cudaInitialIntegrate(Parameter *param, Atom *atom) {
    const int threads_num = 16;
    dim3 block_size = dim3(threads_num, 1, 1);

    #ifdef USE_SUPER_CLUSTERS
    dim3 grid_size = dim3(atom->Nsclusters_local/(threads_num)+1, 1, 1);
    cudaInitialIntegrateSup_warp<<<grid_size, block_size>>>(cuda_scl_x, cuda_scl_v, cuda_scl_f,
                                                            cuda_nclusters,
                                                            cuda_natoms, atom->Nsclusters_local, param->dtforce, param->dt);
    #else
    dim3 grid_size = dim3(atom->Nclusters_local/(threads_num)+1, 1, 1);
    cudaInitialIntegrate_warp<<<grid_size, block_size>>>(cuda_cl_x, cuda_cl_v, cuda_cl_f,
                                                         cuda_natoms, atom->Nclusters_local, param->dtforce, param->dt);
    #endif //USE_SUPER_CLUSTERS
    cuda_assert("cudaInitialIntegrate", hipPeekAtLastError());
    cuda_assert("cudaInitialIntegrate", hipDeviceSynchronize());
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
extern "C"
void cudaUpdatePbc(Atom *atom, Parameter *param) {
    const int threads_num = 512;
    dim3 block_size = dim3(threads_num, 1, 1);;
    dim3 grid_size = dim3(atom->Nclusters_ghost/(threads_num)+1, 1, 1);

#ifdef USE_SUPER_CLUSTERS
    cudaUpdatePbcSup_warp<<<grid_size, block_size>>>(cuda_scl_x, cuda_border_map,
                                       cuda_jclusters_natoms, cuda_PBCx, cuda_PBCy, cuda_PBCz,
                                       atom->Nclusters_local, atom->Nclusters_ghost,
                                       param->xprd, param->yprd, param->zprd);
#else
    cudaUpdatePbc_warp<<<grid_size, block_size>>>(cuda_cl_x, cuda_border_map,
                                                  cuda_jclusters_natoms, cuda_PBCx, cuda_PBCy, cuda_PBCz,
                                                  atom->Nclusters_local, atom->Nclusters_ghost,
                                                  param->xprd, param->yprd, param->zprd);
#endif //USE_SUPER_CLUSTERS
    cuda_assert("cudaUpdatePbc", hipPeekAtLastError());
    cuda_assert("cudaUpdatePbc", hipDeviceSynchronize());
}

extern "C"
double computeForceLJ_cuda(Parameter *param, Atom *atom, Neighbor *neighbor, Stats *stats) {
    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6 = param->sigma6;
    MD_FLOAT epsilon = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    if (isReneighboured) {
        for(int ci = 0; ci < atom->Nclusters_local; ci++) {
            memcpyToGPU(&cuda_numneigh[ci], &neighbor->numneigh[ci], sizeof(int));
            memcpyToGPU(&cuda_neighbors[ci * neighbor->maxneighs], &neighbor->neighbors[ci * neighbor->maxneighs], neighbor->numneigh[ci] * sizeof(int));
        }

        isReneighboured = 0;
    }

    const int threads_num = 1;
    dim3 block_size = dim3(threads_num, CLUSTER_M, CLUSTER_N);
    dim3 grid_size = dim3(atom->Nclusters_local/threads_num+1, 1, 1);
    double S = getTimeStamp();
    LIKWID_MARKER_START("force");
    computeForceLJ_cuda_warp<<<grid_size, block_size>>>(cuda_cl_x, cuda_cl_f,
                                                        atom->Nclusters_local, atom->Nclusters_max,
                                                        cuda_numneigh, cuda_neighbors,
                                                        neighbor->half_neigh, neighbor->maxneighs, cutforcesq,
                                                        sigma6, epsilon);
    cuda_assert("computeForceLJ_cuda", hipPeekAtLastError());
    cuda_assert("computeForceLJ_cuda", hipDeviceSynchronize());
    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    return E-S;
}

extern "C"
void cudaFinalIntegrate(Parameter *param, Atom *atom) {
    const int threads_num = 16;
    dim3 block_size = dim3(threads_num, 1, 1);

    #ifdef USE_SUPER_CLUSTERS
    dim3 grid_size = dim3(atom->Nsclusters_local/(threads_num)+1, 1, 1);
    cudaFinalIntegrateSup_warp<<<grid_size, block_size>>>(cuda_scl_v, cuda_scl_f,
                                                          cuda_nclusters, cuda_natoms,
                                                          atom->Nsclusters_local, param->dt);
    #else
    dim3 grid_size = dim3(atom->Nclusters_local/(threads_num)+1, 1, 1);
    cudaFinalIntegrate_warp<<<grid_size, block_size>>>(cuda_cl_v, cuda_cl_f, cuda_natoms,
                                                          atom->Nclusters_local, param->dt);
    #endif //USE_SUPER_CLUSTERS
    cuda_assert("cudaFinalIntegrate", hipPeekAtLastError());
    cuda_assert("cudaFinalIntegrate", hipDeviceSynchronize());
}
