#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
extern "C" {

#include <stdio.h>
//---
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
//---
#include <likwid-marker.h>
//---
#include <atom.h>
#include <device.h>
#include <neighbor.h>
#include <parameter.h>
#include <stats.h>
#include <timing.h>
#include <util.h>

}

extern "C" {
    MD_FLOAT *cuda_cl_x;
    MD_FLOAT *cuda_cl_v;
    MD_FLOAT *cuda_cl_f;
    int *cuda_neighbors;
    int *cuda_numneigh;
    int *cuda_natoms;
    int *natoms;
    int *ngatoms;
    int *cuda_border_map;
    int *cuda_jclusters_natoms;
    MD_FLOAT *cuda_bbminx, *cuda_bbmaxx;
    MD_FLOAT *cuda_bbminy, *cuda_bbmaxy;
    MD_FLOAT *cuda_bbminz, *cuda_bbmaxz;
    int *cuda_PBCx, *cuda_PBCy, *cuda_PBCz;
    int isReneighboured;
}

extern "C"
void initDevice(Atom *atom, Neighbor *neighbor) {
    cuda_assert("cudaDeviceSetup", hipDeviceReset());
    cuda_assert("cudaDeviceSetup", hipSetDevice(0));
    cuda_cl_x               =   (MD_FLOAT *) allocateGPU(atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_cl_v               =   (MD_FLOAT *) allocateGPU(atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_cl_f               =   (MD_FLOAT *) allocateGPU(atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    cuda_natoms             =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_jclusters_natoms   =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_border_map         =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCx               =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCy               =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_PBCz               =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_numneigh           =   (int *) allocateGPU(atom->Nclusters_max * sizeof(int));
    cuda_neighbors          =   (int *) allocateGPU(atom->Nclusters_max * neighbor->maxneighs * sizeof(int));
    natoms = (int *) malloc(atom->Nclusters_max);
    ngatoms = (int *) malloc(atom->Nclusters_max);
    isReneighboured = 1;
}

extern "C"
void copyDataToCUDADevice(Atom *atom) {
    memcpyToGPU(cuda_cl_x, atom->cl_x, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_cl_v, atom->cl_v, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyToGPU(cuda_cl_f, atom->cl_f, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));

    for(int ci = 0; ci < atom->Nclusters_local; ci++) {
        natoms[ci] = atom->iclusters[ci].natoms;
    }

    memcpyToGPU(cuda_natoms, natoms, atom->Nclusters_local * sizeof(int));

    int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int ncj = atom->Nclusters_local / jfac;
    for(int cg = 0; cg < atom->Nclusters_ghost; cg++) {
        const int cj = ncj + cg;
        ngatoms[cg] = atom->jclusters[cj].natoms;
    }

    memcpyToGPU(cuda_jclusters_natoms, ngatoms, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_border_map, atom->border_map, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCx, atom->PBCx, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCy, atom->PBCy, atom->Nclusters_ghost * sizeof(int));
    memcpyToGPU(cuda_PBCz, atom->PBCz, atom->Nclusters_ghost * sizeof(int));
}

extern "C"
void copyDataFromCUDADevice(Atom *atom) {
    memcpyFromGPU(atom->cl_x, cuda_cl_x, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->cl_v, cuda_cl_v, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    memcpyFromGPU(atom->cl_f, cuda_cl_f, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
}

extern "C"
void cudaDeviceFree() {
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_x));
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_v));
    cuda_assert("cudaDeviceFree", hipFree(cuda_cl_f));
    cuda_assert("cudaDeviceFree", hipFree(cuda_numneigh));
    cuda_assert("cudaDeviceFree", hipFree(cuda_neighbors));
    cuda_assert("cudaDeviceFree", hipFree(cuda_natoms));
    cuda_assert("cudaDeviceFree", hipFree(cuda_border_map));
    cuda_assert("cudaDeviceFree", hipFree(cuda_jclusters_natoms));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCx));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCy));
    cuda_assert("cudaDeviceFree", hipFree(cuda_PBCz));
    free(natoms);
    free(ngatoms);
}

__global__ void cudaInitialIntegrate_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                         int *cuda_natoms,
                                         int Nclusters_local, MD_FLOAT dtforce, MD_FLOAT dt) {

    unsigned int ci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci_pos >= Nclusters_local) return;

    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci_pos);
    MD_FLOAT *ci_x = &cuda_cl_x[ci_vec_base];
    MD_FLOAT *ci_v = &cuda_cl_v[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];

    for (int cii = 0; cii < cuda_natoms[ci_pos]; cii++) {
        ci_v[CL_X_OFFSET + cii] += dtforce * ci_f[CL_X_OFFSET + cii];
        ci_v[CL_Y_OFFSET + cii] += dtforce * ci_f[CL_Y_OFFSET + cii];
        ci_v[CL_Z_OFFSET + cii] += dtforce * ci_f[CL_Z_OFFSET + cii];
        ci_x[CL_X_OFFSET + cii] += dt * ci_v[CL_X_OFFSET + cii];
        ci_x[CL_Y_OFFSET + cii] += dt * ci_v[CL_Y_OFFSET + cii];
        ci_x[CL_Z_OFFSET + cii] += dt * ci_v[CL_Z_OFFSET + cii];
    }
}

__global__ void cudaUpdatePbc_warp(MD_FLOAT *cuda_cl_x, int *cuda_border_map,
                                   int *cuda_jclusters_natoms,
                                   int *cuda_PBCx,
                                   int *cuda_PBCy,
                                   int *cuda_PBCz,
                                   int Nclusters_local,
                                   int Nclusters_ghost,
                                   MD_FLOAT param_xprd,
                                   MD_FLOAT param_yprd,
                                   MD_FLOAT param_zprd) {
    unsigned int cg = blockDim.x * blockIdx.x + threadIdx.x;
    if (cg >= Nclusters_ghost) return;

    int jfac = MAX(1, CLUSTER_N / CLUSTER_M);
    int ncj = Nclusters_local / jfac;
    MD_FLOAT xprd = param_xprd;
    MD_FLOAT yprd = param_yprd;
    MD_FLOAT zprd = param_zprd;

    const int cj = ncj + cg;
    int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
    int bmap_vec_base = CJ_VECTOR_BASE_INDEX(cuda_border_map[cg]);
    MD_FLOAT *cj_x = &cuda_cl_x[cj_vec_base];
    MD_FLOAT *bmap_x = &cuda_cl_x[bmap_vec_base];

    for(int cjj = 0; cjj < cuda_jclusters_natoms[cg]; cjj++) {
        cj_x[CL_X_OFFSET + cjj] = bmap_x[CL_X_OFFSET + cjj] + cuda_PBCx[cg] * xprd;
        cj_x[CL_Y_OFFSET + cjj] = bmap_x[CL_Y_OFFSET + cjj] + cuda_PBCy[cg] * yprd;
        cj_x[CL_Z_OFFSET + cjj] = bmap_x[CL_Z_OFFSET + cjj] + cuda_PBCz[cg] * zprd;
    }
}

__global__ void computeForceLJ_cuda_warp(MD_FLOAT *cuda_cl_x, MD_FLOAT *cuda_cl_f,
                                         int Nclusters_local, int Nclusters_max,
                                         int *cuda_numneigh, int *cuda_neighs, int half_neigh, int maxneighs,
                                         MD_FLOAT cutforcesq, MD_FLOAT sigma6, MD_FLOAT epsilon) {

    unsigned int ci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int cii_pos = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int cjj_pos = blockDim.z * blockIdx.z + threadIdx.z;
    if ((ci_pos >= Nclusters_local) || (cii_pos >= CLUSTER_M) || (cjj_pos >= CLUSTER_N)) return;

    int ci_cj0 = CJ0_FROM_CI(ci_pos);
    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci_pos);
    MD_FLOAT *ci_x = &cuda_cl_x[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];
    int numneighs = cuda_numneigh[ci_pos];
    for(int k = 0; k < numneighs; k++) {
        int cj = (&cuda_neighs[ci_pos * maxneighs])[k];
        int cj_vec_base = CJ_VECTOR_BASE_INDEX(cj);
        MD_FLOAT *cj_x = &cuda_cl_x[cj_vec_base];
        MD_FLOAT *cj_f = &cuda_cl_f[cj_vec_base];

        MD_FLOAT xtmp = ci_x[CL_X_OFFSET + cii_pos];
        MD_FLOAT ytmp = ci_x[CL_Y_OFFSET + cii_pos];
        MD_FLOAT ztmp = ci_x[CL_Z_OFFSET + cii_pos];
        MD_FLOAT fix = 0;
        MD_FLOAT fiy = 0;
        MD_FLOAT fiz = 0;

        int cond;
#if CLUSTER_M == CLUSTER_N
        cond = half_neigh ? (ci_cj0 != cj || cii_pos < cjj_pos) :
               (ci_cj0 != cj || cii_pos != cjj_pos);
#elif CLUSTER_M < CLUSTER_N
        cond = half_neigh ? (ci_cj0 != cj || cii_pos + CLUSTER_M * (ci_pos & 0x1) < cjj_pos) :
                                            (ci_cj0 != cj || cii_pos + CLUSTER_M * (ci_pos & 0x1) != cjj_pos);
#endif
        if(cond) {
            MD_FLOAT delx = xtmp - cj_x[CL_X_OFFSET + cjj_pos];
            MD_FLOAT dely = ytmp - cj_x[CL_Y_OFFSET + cjj_pos];
            MD_FLOAT delz = ztmp - cj_x[CL_Z_OFFSET + cjj_pos];
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;
            if(rsq < cutforcesq) {
                MD_FLOAT sr2 = 1.0 / rsq;
                MD_FLOAT sr6 = sr2 * sr2 * sr2 * sigma6;
                MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;

                if(half_neigh) {
                    atomicAdd(&cj_f[CL_X_OFFSET + cjj_pos], -delx * force);
                    atomicAdd(&cj_f[CL_Y_OFFSET + cjj_pos], -dely * force);
                    atomicAdd(&cj_f[CL_Z_OFFSET + cjj_pos], -delz * force);
                }

                fix += delx * force;
                fiy += dely * force;
                fiz += delz * force;

                atomicAdd(&ci_f[CL_X_OFFSET + cii_pos], fix);
                atomicAdd(&ci_f[CL_Y_OFFSET + cii_pos], fiy);
                atomicAdd(&ci_f[CL_Z_OFFSET + cii_pos], fiz);
            }
        }
    }
}

__global__ void cudaFinalIntegrate_warp(MD_FLOAT *cuda_cl_v, MD_FLOAT *cuda_cl_f,
                                          int *cuda_natoms,
                                          int Nclusters_local, MD_FLOAT dtforce) {

    unsigned int ci_pos = blockDim.x * blockIdx.x + threadIdx.x;
    if (ci_pos >= Nclusters_local) return;

    int ci_vec_base = CI_VECTOR_BASE_INDEX(ci_pos);
    MD_FLOAT *ci_v = &cuda_cl_v[ci_vec_base];
    MD_FLOAT *ci_f = &cuda_cl_f[ci_vec_base];

    for (int cii = 0; cii < cuda_natoms[ci_pos]; cii++) {
        ci_v[CL_X_OFFSET + cii] += dtforce * ci_f[CL_X_OFFSET + cii];
        ci_v[CL_Y_OFFSET + cii] += dtforce * ci_f[CL_Y_OFFSET + cii];
        ci_v[CL_Z_OFFSET + cii] += dtforce * ci_f[CL_Z_OFFSET + cii];
    }
}

extern "C"
void cudaInitialIntegrate(Parameter *param, Atom *atom) {
    const int threads_num = 16;
    dim3 block_size = dim3(threads_num, 1, 1);
    dim3 grid_size = dim3(atom->Nclusters_local/(threads_num)+1, 1, 1);
    cudaInitialIntegrate_warp<<<grid_size, block_size>>>(cuda_cl_x, cuda_cl_v, cuda_cl_f,
                                                         cuda_natoms, atom->Nclusters_local, param->dtforce, param->dt);
    cuda_assert("cudaInitialIntegrate", hipPeekAtLastError());
    cuda_assert("cudaInitialIntegrate", hipDeviceSynchronize());
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
extern "C"
void cudaUpdatePbc(Atom *atom, Parameter *param) {
    const int threads_num = 512;
    dim3 block_size = dim3(threads_num, 1, 1);;
    dim3 grid_size = dim3(atom->Nclusters_ghost/(threads_num)+1, 1, 1);;
    cudaUpdatePbc_warp<<<grid_size, block_size>>>(cuda_cl_x, cuda_border_map,
                                       cuda_jclusters_natoms, cuda_PBCx, cuda_PBCy, cuda_PBCz,
                                       atom->Nclusters_local, atom->Nclusters_ghost,
                                       param->xprd, param->yprd, param->zprd);
    cuda_assert("cudaUpdatePbc", hipPeekAtLastError());
    cuda_assert("cudaUpdatePbc", hipDeviceSynchronize());
}

extern "C"
double computeForceLJ_cuda(Parameter *param, Atom *atom, Neighbor *neighbor, Stats *stats) {
    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6 = param->sigma6;
    MD_FLOAT epsilon = param->epsilon;

    memsetGPU(cuda_cl_f, 0, atom->Nclusters_max * CLUSTER_M * 3 * sizeof(MD_FLOAT));
    if (isReneighboured) {
        for(int ci = 0; ci < atom->Nclusters_local; ci++) {
            memcpyToGPU(&cuda_numneigh[ci], &neighbor->numneigh[ci], sizeof(int));
            memcpyToGPU(&cuda_neighbors[ci * neighbor->maxneighs], &neighbor->neighbors[ci * neighbor->maxneighs], neighbor->numneigh[ci] * sizeof(int));
        }

        isReneighboured = 0;
    }

    const int threads_num = 1;
    dim3 block_size = dim3(threads_num, CLUSTER_M, CLUSTER_N);
    dim3 grid_size = dim3(atom->Nclusters_local/threads_num+1, 1, 1);
    double S = getTimeStamp();
    LIKWID_MARKER_START("force");
    computeForceLJ_cuda_warp<<<grid_size, block_size>>>(cuda_cl_x, cuda_cl_f,
                                                        atom->Nclusters_local, atom->Nclusters_max,
                                                        cuda_numneigh, cuda_neighbors,
                                                        neighbor->half_neigh, neighbor->maxneighs, cutforcesq,
                                                        sigma6, epsilon);
    cuda_assert("computeForceLJ_cuda", hipPeekAtLastError());
    cuda_assert("computeForceLJ_cuda", hipDeviceSynchronize());
    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    return E-S;
}

extern "C"
void cudaFinalIntegrate(Parameter *param, Atom *atom) {
    const int threads_num = 16;
    dim3 block_size = dim3(threads_num, 1, 1);
    dim3 grid_size = dim3(atom->Nclusters_local/(threads_num)+1, 1, 1);
    cudaFinalIntegrate_warp<<<grid_size, block_size>>>(cuda_cl_v, cuda_cl_f, cuda_natoms, atom->Nclusters_local, param->dt);
    cuda_assert("cudaFinalIntegrate", hipPeekAtLastError());
    cuda_assert("cudaFinalIntegrate", hipDeviceSynchronize());
}
