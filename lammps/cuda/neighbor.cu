#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2021 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
//---

extern "C" {

#include <atom.h>
#include <cuda_atom.h>
#include <parameter.h>
#include <neighbor.h>
#include <util.h>

}

extern MD_FLOAT xprd, yprd, zprd;
extern MD_FLOAT bininvx, bininvy, bininvz;
extern int mbinxlo, mbinylo, mbinzlo;
extern int nbinx, nbiny, nbinz;
extern int mbinx, mbiny, mbinz; // n bins in x, y, z
extern int mbins; //total number of bins
extern int atoms_per_bin;  // max atoms per bin
extern MD_FLOAT cutneighsq;  // neighbor cutoff squared
extern int nmax;
extern int nstencil;      // # of bins in stencil
extern int* stencil;      // stencil list of bin offsets
static int* c_stencil = NULL;
static int* c_resize_needed = NULL;
static int* c_new_maxneighs = NULL;
static Binning c_binning {
    .bincount = NULL,
    .bins = NULL,
    .mbins = 0,
    .atoms_per_bin = 0
};

__device__ int coord2bin_device(MD_FLOAT xin, MD_FLOAT yin, MD_FLOAT zin, Neighbor_params np) {
    int ix, iy, iz;

    if(xin >= np.xprd) {
        ix = (int)((xin - np.xprd) * np.bininvx) + np.nbinx - np.mbinxlo;
    } else if(xin >= 0.0) {
        ix = (int)(xin * np.bininvx) - np.mbinxlo;
    } else {
        ix = (int)(xin * np.bininvx) - np.mbinxlo - 1;
    }

    if(yin >= np.yprd) {
        iy = (int)((yin - np.yprd) * np.bininvy) + np.nbiny - np.mbinylo;
    } else if(yin >= 0.0) {
        iy = (int)(yin * np.bininvy) - np.mbinylo;
    } else {
        iy = (int)(yin * np.bininvy) - np.mbinylo - 1;
    }

    if(zin >= np.zprd) {
        iz = (int)((zin - np.zprd) * np.bininvz) + np.nbinz - np.mbinzlo;
    } else if(zin >= 0.0) {
        iz = (int)(zin * np.bininvz) - np.mbinzlo;
    } else {
        iz = (int)(zin * np.bininvz) - np.mbinzlo - 1;
    }

    return (iz * np.mbiny * np.mbinx + iy * np.mbinx + ix + 1);
}

/* sorts the contents of a bin to make it comparable to the CPU version */
/* uses bubble sort since atoms per bin should be relatively small and can be done in situ */
__global__ void sort_bin_contents_kernel(int* bincount, int* bins, int mbins, int atoms_per_bin){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= mbins) {
        return;
    }

    int atoms_in_bin = bincount[i];
    int *bin_ptr = &bins[i * atoms_per_bin];
    int sorted;
    do {
        sorted = 1;
        int tmp;
        for(int index = 0; index < atoms_in_bin - 1; index++){
            if (bin_ptr[index] > bin_ptr[index + 1]){
                tmp = bin_ptr[index];
                bin_ptr[index] = bin_ptr[index + 1];
                bin_ptr[index + 1] = tmp;
                sorted = 0;
            }
        }
    } while (!sorted);
}

__global__ void binatoms_kernel(Atom a, int* bincount, int* bins, int atoms_per_bin, Neighbor_params np, int *resize_needed) {
    Atom* atom = &a;
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    int nall = atom->Nlocal + atom->Nghost;
    if(i >= nall){
        return;
    }
    
    MD_FLOAT x = atom_x(i);
    MD_FLOAT y = atom_y(i);
    MD_FLOAT z = atom_z(i);
    int ibin = coord2bin_device(x, y, z, np);
    int ac = atomicAdd(&bincount[ibin], 1);
            
    if(ac < atoms_per_bin){
        bins[ibin * atoms_per_bin + ac] = i;
    } else {
        atomicMax(resize_needed, ac);
    }
}

__global__ void compute_neighborhood(Atom a, Neighbor neigh, Neighbor_params np, int nstencil, int* stencil,
                                     int* bins, int atoms_per_bin, int *bincount, int *new_maxneighs, MD_FLOAT cutneighsq) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nlocal = a.Nlocal;
    if( i >= Nlocal ) {
        return;
    }
    
    Atom *atom = &a;
    Neighbor *neighbor = &neigh;
    
    int* neighptr = &(neighbor->neighbors[i]);
    int n = 0;
    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);
    int ibin = coord2bin_device(xtmp, ytmp, ztmp, np);
#ifdef EXPLICIT_TYPES
    int type_i = atom->type[i];
#endif
    for(int k = 0; k < nstencil; k++) {
        int jbin = ibin + stencil[k];
        int* loc_bin = &bins[jbin * atoms_per_bin];

        for(int m = 0; m < bincount[jbin]; m++) {
            int j = loc_bin[m];

            if ( j == i ){
                continue;
            }

            MD_FLOAT delx = xtmp - atom_x(j);
            MD_FLOAT dely = ytmp - atom_y(j);
            MD_FLOAT delz = ztmp - atom_z(j);
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

#ifdef EXPLICIT_TYPES
            int type_j = atom->type[j];
            const MD_FLOAT cutoff = atom->cutneighsq[type_i * atom->ntypes + type_j];
#else
            const MD_FLOAT cutoff = cutneighsq;
#endif

            if( rsq <= cutoff ) {
                int idx = atom->Nlocal * n;
                neighptr[idx] = j;
                n += 1;
            }
        }
    }

    neighbor->numneigh[i] = n;
    if(n > neighbor->maxneighs) {
        atomicMax(new_maxneighs, n);
    }
}

void binatoms_cuda(Atom *c_atom, Binning *c_binning, int *c_resize_needed, Neighbor_params *np, const int threads_per_block) {
    int nall = c_atom->Nlocal + c_atom->Nghost;
    int resize = 1;
    const int num_blocks = ceil((float) nall / (float) threads_per_block);

    while(resize > 0) {
        resize = 0;
        checkCUDAError("binatoms_cuda c_binning->bincount memset", hipMemset(c_binning->bincount, 0, c_binning->mbins * sizeof(int)));
        checkCUDAError("binatoms_cuda c_resize_needed memset", hipMemset(c_resize_needed, 0, sizeof(int)) );

        /*binatoms_kernel(Atom a, int* bincount, int* bins, int c_binning->atoms_per_bin, Neighbor_params np, int *resize_needed) */
        binatoms_kernel<<<num_blocks, threads_per_block>>>(*c_atom, c_binning->bincount, c_binning->bins, c_binning->atoms_per_bin, *np, c_resize_needed);

	    checkCUDAError( "PeekAtLastError binatoms kernel", hipPeekAtLastError() );
	    checkCUDAError( "DeviceSync binatoms kernel", hipDeviceSynchronize() );
        
	    checkCUDAError("binatoms_cuda c_resize_needed memcpy back", hipMemcpy(&resize, c_resize_needed, sizeof(int), hipMemcpyDeviceToHost) );

        if(resize) {
            hipFree(c_binning->bins);
            c_binning->atoms_per_bin *= 2;
            checkCUDAError("binatoms_cuda c_binning->bins resize malloc", hipMalloc(&c_binning->bins, c_binning->mbins * c_binning->atoms_per_bin * sizeof(int)) );
        }
    }

    atoms_per_bin = c_binning->atoms_per_bin;
    const int sortBlocks = ceil((float)mbins / (float)threads_per_block);
    /*void sort_bin_contents_kernel(int* bincount, int* bins, int mbins, int atoms_per_bin)*/
    sort_bin_contents_kernel<<<sortBlocks, threads_per_block>>>(c_binning->bincount, c_binning->bins, c_binning->mbins, c_binning->atoms_per_bin);
    checkCUDAError( "PeekAtLastError sort_bin_contents kernel", hipPeekAtLastError() );
    checkCUDAError( "DeviceSync sort_bin_contents kernel", hipDeviceSynchronize() );
}

void buildNeighbor_cuda(Atom *atom, Neighbor *neighbor, Atom *c_atom, Neighbor *c_neighbor) {
    const int num_threads_per_block = get_num_threads();
    int nall = atom->Nlocal + atom->Nghost;
    c_neighbor->maxneighs = neighbor->maxneighs;

    hipProfilerStart();
    /* upload stencil */
    // TODO move all of this initialization into its own method
    if(c_stencil == NULL){
        checkCUDAError( "buildNeighbor c_n_stencil malloc", hipMalloc((void**)&c_stencil, nstencil * sizeof(int)) );
        checkCUDAError( "buildNeighbor c_n_stencil memcpy", hipMemcpy(c_stencil, stencil, nstencil * sizeof(int), hipMemcpyHostToDevice ));
    }

    if(c_binning.mbins == 0){
        c_binning.mbins = mbins;
        c_binning.atoms_per_bin = atoms_per_bin;
        checkCUDAError( "buildNeighbor c_binning->bincount malloc", hipMalloc((void**)&(c_binning.bincount), c_binning.mbins * sizeof(int)) );
        checkCUDAError( "buidlNeighbor c_binning->bins malloc", hipMalloc((void**)&(c_binning.bins), c_binning.mbins * c_binning.atoms_per_bin * sizeof(int)) );
    }

    Neighbor_params np {
        .xprd = xprd,
        .yprd = yprd,
        .zprd = zprd,
        .bininvx = bininvx,
        .bininvy = bininvy,
        .bininvz = bininvz,
        .mbinxlo = mbinxlo,
        .mbinylo = mbinylo,
        .mbinzlo = mbinzlo,
        .nbinx = nbinx,
        .nbiny = nbiny,
        .nbinz = nbinz,
        .mbinx = mbinx,
        .mbiny = mbiny,
        .mbinz = mbinz
    };

    if(c_resize_needed == NULL){
        checkCUDAError("buildNeighbor c_resize_needed malloc", hipMalloc((void**)&c_resize_needed, sizeof(int)) );
    }

    /* bin local & ghost atoms */
    binatoms_cuda(c_atom, &c_binning, c_resize_needed, &np, num_threads_per_block);
    if(c_new_maxneighs == NULL){
        checkCUDAError("c_new_maxneighs malloc", hipMalloc((void**)&c_new_maxneighs, sizeof(int) ));
    }

    int resize = 1;
    
    /* extend c_neighbor arrays if necessary */
    if(nall > nmax) {
        nmax = nall;
        if(c_neighbor->numneigh) hipFree(c_neighbor->numneigh);
        if(c_neighbor->neighbors) hipFree(c_neighbor->neighbors);
        checkCUDAError( "buildNeighbor c_numneigh malloc", hipMalloc((void**)&(c_neighbor->numneigh), nmax * sizeof(int)) );
        checkCUDAError( "buildNeighbor c_neighbors malloc", hipMalloc((void**)&(c_neighbor->neighbors), nmax * c_neighbor->maxneighs * sizeof(int)) );
    }

    /* loop over each atom, storing neighbors */
    while(resize) {
        resize = 0;

        checkCUDAError("c_new_maxneighs memset", hipMemset(c_new_maxneighs, 0, sizeof(int) ));

        // TODO call compute_neigborhood kernel here
        const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
        /*compute_neighborhood(Atom a, Neighbor neigh, Neighbor_params np, int nstencil, int* stencil,
                                     int* bins, int atoms_per_bin, int *bincount, int *new_maxneighs)
         * */
        compute_neighborhood<<<num_blocks, num_threads_per_block>>>(*c_atom, *c_neighbor,
                                                                    np, nstencil, c_stencil,
                                                                    c_binning.bins, c_binning.atoms_per_bin, c_binning.bincount,
                                                                    c_new_maxneighs,
								                                    cutneighsq);

        checkCUDAError( "PeekAtLastError ComputeNeighbor", hipPeekAtLastError() );
        checkCUDAError( "DeviceSync ComputeNeighbor", hipDeviceSynchronize() );

        // TODO copy the value of c_new_maxneighs back to host and check if it has been modified
        int new_maxneighs;
        checkCUDAError("c_new_maxneighs memcpy back", hipMemcpy(&new_maxneighs, c_new_maxneighs, sizeof(int), hipMemcpyDeviceToHost));
        if (new_maxneighs > c_neighbor->maxneighs){
            resize = 1;
        }

        if(resize) {
            printf("RESIZE %d\n", c_neighbor->maxneighs);
            c_neighbor->maxneighs = new_maxneighs * 1.2;
            printf("NEW SIZE %d\n", c_neighbor->maxneighs);
            hipFree(c_neighbor->neighbors);
            checkCUDAError("c_neighbor->neighbors resize malloc", hipMalloc((void**)(&c_neighbor->neighbors), c_atom->Nmax * c_neighbor->maxneighs * sizeof(int)));
        }

    }

    neighbor->maxneighs = c_neighbor->maxneighs;
    hipProfilerStop();
}
