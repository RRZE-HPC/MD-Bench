#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2021 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
//---

extern "C" {

#include <atom.h>
#include <cuda_atom.h>
#include <parameter.h>
#include <neighbor.h>
#include <util.h>

}

extern MD_FLOAT xprd, yprd, zprd;
extern MD_FLOAT bininvx, bininvy, bininvz;
extern int mbinxlo, mbinylo, mbinzlo;
extern int nbinx, nbiny, nbinz;
extern int mbinx, mbiny, mbinz; // n bins in x, y, z
extern int mbins; //total number of bins
extern int atoms_per_bin;  // max atoms per bin
extern MD_FLOAT cutneighsq;  // neighbor cutoff squared
extern int nmax;
extern int nstencil;      // # of bins in stencil
extern int* stencil;      // stencil list of bin offsets
static int* c_stencil = NULL;
static int* c_resize_needed = NULL;
static int* c_new_maxneighs = NULL;
static Binning c_binning {
    .bincount = NULL,
    .bins = NULL,
    .mbins = 0,
    .atoms_per_bin = 0
};

__device__ int coord2bin_device(MD_FLOAT xin, MD_FLOAT yin, MD_FLOAT zin, Neighbor_params np) {
    int ix, iy, iz;

    if(xin >= np.xprd) {
        ix = (int)((xin - np.xprd) * np.bininvx) + np.nbinx - np.mbinxlo;
    } else if(xin >= 0.0) {
        ix = (int)(xin * np.bininvx) - np.mbinxlo;
    } else {
        ix = (int)(xin * np.bininvx) - np.mbinxlo - 1;
    }

    if(yin >= np.yprd) {
        iy = (int)((yin - np.yprd) * np.bininvy) + np.nbiny - np.mbinylo;
    } else if(yin >= 0.0) {
        iy = (int)(yin * np.bininvy) - np.mbinylo;
    } else {
        iy = (int)(yin * np.bininvy) - np.mbinylo - 1;
    }

    if(zin >= np.zprd) {
        iz = (int)((zin - np.zprd) * np.bininvz) + np.nbinz - np.mbinzlo;
    } else if(zin >= 0.0) {
        iz = (int)(zin * np.bininvz) - np.mbinzlo;
    } else {
        iz = (int)(zin * np.bininvz) - np.mbinzlo - 1;
    }

    return (iz * np.mbiny * np.mbinx + iy * np.mbinx + ix + 1);
}

/* sorts the contents of a bin to make it comparable to the CPU version */
/* uses bubble sort since atoms per bin should be relatively small and can be done in situ */
__global__ void sort_bin_contents_kernel(int* bincount, int* bins, int mbins, int atoms_per_bin){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= mbins) {
        return;
    }

    int atoms_in_bin = bincount[i];
    int *bin_ptr = &bins[i * atoms_per_bin];
    int sorted;
    do {
        sorted = 1;
        int tmp;
        for(int index = 0; index < atoms_in_bin - 1; index++){
            if (bin_ptr[index] > bin_ptr[index + 1]){
                tmp = bin_ptr[index];
                bin_ptr[index] = bin_ptr[index + 1];
                bin_ptr[index + 1] = tmp;
                sorted = 0;
            }
        }
    } while (!sorted);
}

__global__ void binatoms_kernel(Atom a, int* bincount, int* bins, int atoms_per_bin, Neighbor_params np, int *resize_needed) {
    Atom* atom = &a;
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    int nall = atom->Nlocal + atom->Nghost;
    if(i >= nall){
        return;
    }
    
    MD_FLOAT x = atom_x(i);
    MD_FLOAT y = atom_y(i);
    MD_FLOAT z = atom_z(i);
    int ibin = coord2bin_device(x, y, z, np);
    int ac = atomicAdd(&bincount[ibin], 1);
            
    if(ac < atoms_per_bin){
        bins[ibin * atoms_per_bin + ac] = i;
    } else {
        atomicMax(resize_needed, ac);
    }
}

__global__ void compute_neighborhood(Atom a, Neighbor neigh, Neighbor_params np, int nstencil, int* stencil,
                                     int* bins, int atoms_per_bin, int *bincount, int *new_maxneighs, MD_FLOAT cutneighsq) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nlocal = a.Nlocal;
    if( i >= Nlocal ) {
        return;
    }
    
    Atom *atom = &a;
    Neighbor *neighbor = &neigh;
    
    int* neighptr = &(neighbor->neighbors[i]);
    int n = 0;
    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);
    int ibin = coord2bin_device(xtmp, ytmp, ztmp, np);
#ifdef EXPLICIT_TYPES
    int type_i = atom->type[i];
#endif
    for(int k = 0; k < nstencil; k++) {
        int jbin = ibin + stencil[k];
        int* loc_bin = &bins[jbin * atoms_per_bin];

        for(int m = 0; m < bincount[jbin]; m++) {
            int j = loc_bin[m];

            if ( j == i ){
                continue;
            }

            MD_FLOAT delx = xtmp - atom_x(j);
            MD_FLOAT dely = ytmp - atom_y(j);
            MD_FLOAT delz = ztmp - atom_z(j);
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

#ifdef EXPLICIT_TYPES
            int type_j = atom->type[j];
            const MD_FLOAT cutoff = atom->cutneighsq[type_i * atom->ntypes + type_j];
#else
            const MD_FLOAT cutoff = cutneighsq;
#endif

            if( rsq <= cutoff ) {
                int idx = atom->Nlocal * n;
                neighptr[idx] = j;
                n += 1;
            }
        }
    }

    neighbor->numneigh[i] = n;
    if(n > neighbor->maxneighs) {
        atomicMax(new_maxneighs, n);
    }
}

void binatoms_cuda(Atom *c_atom, Binning *c_binning, int *c_resize_needed, Neighbor_params *np, const int threads_per_block) {
    int nall = c_atom->Nlocal + c_atom->Nghost;
    int resize = 1;
    const int num_blocks = ceil((float) nall / (float) threads_per_block);

    while(resize > 0) {
        resize = 0;
        memsetGPU(c_binning->bincount, 0, c_binning->mbins * sizeof(int));
        memsetGPU(c_resize_needed, 0, sizeof(int));

        binatoms_kernel<<<num_blocks, threads_per_block>>>(*c_atom, c_binning->bincount, c_binning->bins, c_binning->atoms_per_bin, *np, c_resize_needed);
	    cuda_assert("binatoms", hipPeekAtLastError());
	    cuda_assert("binatoms", hipDeviceSynchronize());

        memcpyFromGPU(&resize, c_resize_needed, sizeof(int));
        if(resize) {
            c_binning->atoms_per_bin *= 2;
            c_binning->bins = (int *) reallocateGPU(c_binning->bins, c_binning->mbins * c_binning->atoms_per_bin * sizeof(int));
        }
    }

    atoms_per_bin = c_binning->atoms_per_bin;
    const int sortBlocks = ceil((float) mbins / (float) threads_per_block);
    sort_bin_contents_kernel<<<sortBlocks, threads_per_block>>>(c_binning->bincount, c_binning->bins, c_binning->mbins, c_binning->atoms_per_bin);
	cuda_assert("sort_bin", hipPeekAtLastError());
	cuda_assert("sort_bin", hipDeviceSynchronize());
}

void buildNeighbor_cuda(Atom *atom, Neighbor *neighbor, Atom *c_atom, Neighbor *c_neighbor) {
    const int num_threads_per_block = get_num_threads();
    int nall = atom->Nlocal + atom->Nghost;
    c_neighbor->maxneighs = neighbor->maxneighs;

    hipProfilerStart();

    // TODO move all of this initialization into its own method
    if(c_stencil == NULL) {
        c_stencil = (int *) allocateGPU(nstencil * sizeof(int));
        memcpyToGPU(c_stencil, stencil, nstencil * sizeof(int));
    }

    if(c_binning.mbins == 0) {
        c_binning.mbins = mbins;
        c_binning.atoms_per_bin = atoms_per_bin;
        c_binning.bincount = (int *) allocateGPU(c_binning.mbins * sizeof(int));
        c_binning.bins = (int *) allocateGPU(c_binning.mbins * c_binning.atoms_per_bin * sizeof(int));
    }

    Neighbor_params np {
        .xprd = xprd,
        .yprd = yprd,
        .zprd = zprd,
        .bininvx = bininvx,
        .bininvy = bininvy,
        .bininvz = bininvz,
        .mbinxlo = mbinxlo,
        .mbinylo = mbinylo,
        .mbinzlo = mbinzlo,
        .nbinx = nbinx,
        .nbiny = nbiny,
        .nbinz = nbinz,
        .mbinx = mbinx,
        .mbiny = mbiny,
        .mbinz = mbinz
    };

    if(c_resize_needed == NULL) {
        c_resize_needed = (int *) allocateGPU(sizeof(int));
    }

    /* bin local & ghost atoms */
    binatoms_cuda(c_atom, &c_binning, c_resize_needed, &np, num_threads_per_block);
    if(c_new_maxneighs == NULL) {
        c_new_maxneighs = (int *) allocateGPU(sizeof(int));
    }

    int resize = 1;
    
    /* extend c_neighbor arrays if necessary */
    if(nall > nmax) {
        nmax = nall;
        c_neighbor->neighbors = (int *) reallocateGPU(c_neighbor->neighbors, nmax * c_neighbor->maxneighs * sizeof(int));
        c_neighbor->numneigh  = (int *) reallocateGPU(c_neighbor->numneigh,  nmax * sizeof(int));
    }

    /* loop over each atom, storing neighbors */
    while(resize) {
        resize = 0;
        memsetGPU(c_new_maxneighs, 0, sizeof(int));
        const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
        compute_neighborhood<<<num_blocks, num_threads_per_block>>>(*c_atom, *c_neighbor,
                                                                    np, nstencil, c_stencil,
                                                                    c_binning.bins, c_binning.atoms_per_bin, c_binning.bincount,
                                                                    c_new_maxneighs,
								                                    cutneighsq);

        cuda_assert("compute_neighborhood", hipPeekAtLastError());
        cuda_assert("compute_neighborhood", hipDeviceSynchronize());

        int new_maxneighs;
        memcpyFromGPU(&new_maxneighs, c_new_maxneighs, sizeof(int));
        if (new_maxneighs > c_neighbor->maxneighs){
            resize = 1;
        }

        if(resize) {
            printf("RESIZE %d\n", c_neighbor->maxneighs);
            c_neighbor->maxneighs = new_maxneighs * 1.2;
            printf("NEW SIZE %d\n", c_neighbor->maxneighs);
            c_neighbor->neighbors = (int *) reallocateGPU(c_neighbor->neighbors, c_atom->Nmax * c_neighbor->maxneighs * sizeof(int));
        }

    }

    neighbor->maxneighs = c_neighbor->maxneighs;
    hipProfilerStop();
}
