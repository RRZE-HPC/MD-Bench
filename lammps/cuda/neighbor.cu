#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
//---

extern "C" {

#include <atom.h>
#include <device.h>
#include <parameter.h>
#include <neighbor.h>
#include <util.h>

}

extern MD_FLOAT xprd, yprd, zprd;
extern MD_FLOAT bininvx, bininvy, bininvz;
extern int mbinxlo, mbinylo, mbinzlo;
extern int nbinx, nbiny, nbinz;
extern int mbinx, mbiny, mbinz; // n bins in x, y, z
extern int mbins; //total number of bins
extern int atoms_per_bin;  // max atoms per bin
extern MD_FLOAT cutneighsq;  // neighbor cutoff squared
extern int nmax;
extern int nstencil;      // # of bins in stencil
extern int* stencil;      // stencil list of bin offsets
static int* c_stencil = NULL;
static int* c_resize_needed = NULL;
static int* c_new_maxneighs = NULL;
static Binning c_binning {
    .bincount = NULL,
    .bins = NULL,
    .mbins = 0,
    .atoms_per_bin = 0
};

__device__ int coord2bin_device(MD_FLOAT xin, MD_FLOAT yin, MD_FLOAT zin, Neighbor_params np) {
    int ix, iy, iz;

    if(xin >= np.xprd) {
        ix = (int)((xin - np.xprd) * np.bininvx) + np.nbinx - np.mbinxlo;
    } else if(xin >= 0.0) {
        ix = (int)(xin * np.bininvx) - np.mbinxlo;
    } else {
        ix = (int)(xin * np.bininvx) - np.mbinxlo - 1;
    }

    if(yin >= np.yprd) {
        iy = (int)((yin - np.yprd) * np.bininvy) + np.nbiny - np.mbinylo;
    } else if(yin >= 0.0) {
        iy = (int)(yin * np.bininvy) - np.mbinylo;
    } else {
        iy = (int)(yin * np.bininvy) - np.mbinylo - 1;
    }

    if(zin >= np.zprd) {
        iz = (int)((zin - np.zprd) * np.bininvz) + np.nbinz - np.mbinzlo;
    } else if(zin >= 0.0) {
        iz = (int)(zin * np.bininvz) - np.mbinzlo;
    } else {
        iz = (int)(zin * np.bininvz) - np.mbinzlo - 1;
    }

    return (iz * np.mbiny * np.mbinx + iy * np.mbinx + ix + 1);
}

/* sorts the contents of a bin to make it comparable to the CPU version */
/* uses bubble sort since atoms per bin should be relatively small and can be done in situ */
__global__ void sort_bin_contents_kernel(int* bincount, int* bins, int mbins, int atoms_per_bin){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= mbins) {
        return;
    }

    int atoms_in_bin = bincount[i];
    int *bin_ptr = &bins[i * atoms_per_bin];
    int sorted;
    do {
        sorted = 1;
        int tmp;
        for(int index = 0; index < atoms_in_bin - 1; index++){
            if (bin_ptr[index] > bin_ptr[index + 1]){
                tmp = bin_ptr[index];
                bin_ptr[index] = bin_ptr[index + 1];
                bin_ptr[index + 1] = tmp;
                sorted = 0;
            }
        }
    } while (!sorted);
}

__global__ void binatoms_kernel(DeviceAtom a, int nall, int* bincount, int* bins, int atoms_per_bin, Neighbor_params np, int *resize_needed) {
    DeviceAtom* atom = &a;
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= nall) {
        return;
    }

    MD_FLOAT x = atom_x(i);
    MD_FLOAT y = atom_y(i);
    MD_FLOAT z = atom_z(i);
    int ibin = coord2bin_device(x, y, z, np);
    int ac = atomicAdd(&bincount[ibin], 1);

    if(ac < atoms_per_bin){
        bins[ibin * atoms_per_bin + ac] = i;
    } else {
        atomicMax(resize_needed, ac);
    }
}

__global__ void compute_neighborhood(
    DeviceAtom a, DeviceNeighbor neigh, Neighbor_params np, int nlocal, int maxneighs, int nstencil, int* stencil,
    int* bins, int atoms_per_bin, int *bincount, int *new_maxneighs, MD_FLOAT cutneighsq, int ntypes) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= nlocal) {
        return;
    }

    DeviceAtom *atom = &a;
    DeviceNeighbor *neighbor = &neigh;

    int* neighptr = &(neighbor->neighbors[i]);
    int n = 0;
    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);
    int ibin = coord2bin_device(xtmp, ytmp, ztmp, np);
#ifdef EXPLICIT_TYPES
    int type_i = atom->type[i];
#endif
    for(int k = 0; k < nstencil; k++) {
        int jbin = ibin + stencil[k];
        int* loc_bin = &bins[jbin * atoms_per_bin];

        for(int m = 0; m < bincount[jbin]; m++) {
            int j = loc_bin[m];

            if ( j == i ){
                continue;
            }

            MD_FLOAT delx = xtmp - atom_x(j);
            MD_FLOAT dely = ytmp - atom_y(j);
            MD_FLOAT delz = ztmp - atom_z(j);
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

#ifdef EXPLICIT_TYPES
            int type_j = atom->type[j];
            const MD_FLOAT cutoff = atom->cutneighsq[type_i * ntypes + type_j];
#else
            const MD_FLOAT cutoff = cutneighsq;
#endif

            if( rsq <= cutoff ) {
                int idx = nlocal * n;
                neighptr[idx] = j;
                n += 1;
            }
        }
    }

    neighbor->numneigh[i] = n;
    if(n > maxneighs) {
        atomicMax(new_maxneighs, n);
    }
}

void binatoms_cuda(Atom *atom, Binning *c_binning, int *c_resize_needed, Neighbor_params *np, const int threads_per_block) {
    int nall = atom->Nlocal + atom->Nghost;
    int resize = 1;
    const int num_blocks = ceil((float) nall / (float) threads_per_block);

    while(resize > 0) {
        resize = 0;
        memsetGPU(c_binning->bincount, 0, c_binning->mbins * sizeof(int));
        memsetGPU(c_resize_needed, 0, sizeof(int));

        binatoms_kernel<<<num_blocks, threads_per_block>>>(atom->d_atom, atom->Nlocal + atom->Nghost, c_binning->bincount, c_binning->bins, c_binning->atoms_per_bin, *np, c_resize_needed);
	    cuda_assert("binatoms", hipPeekAtLastError());
	    cuda_assert("binatoms", hipDeviceSynchronize());

        memcpyFromGPU(&resize, c_resize_needed, sizeof(int));
        if(resize) {
            c_binning->atoms_per_bin *= 2;
            c_binning->bins = (int *) reallocateGPU(c_binning->bins, c_binning->mbins * c_binning->atoms_per_bin * sizeof(int));
        }
    }

    atoms_per_bin = c_binning->atoms_per_bin;
    const int sortBlocks = ceil((float) mbins / (float) threads_per_block);
    sort_bin_contents_kernel<<<sortBlocks, threads_per_block>>>(c_binning->bincount, c_binning->bins, c_binning->mbins, c_binning->atoms_per_bin);
	cuda_assert("sort_bin", hipPeekAtLastError());
	cuda_assert("sort_bin", hipDeviceSynchronize());
}

void buildNeighbor_cuda(Atom *atom, Neighbor *neighbor) {
    DeviceNeighbor *d_neighbor = &(neighbor->d_neighbor);
    const int num_threads_per_block = get_cuda_num_threads();
    int nall = atom->Nlocal + atom->Nghost;

    hipProfilerStart();

    // TODO move all of this initialization into its own method
    if(c_stencil == NULL) {
        c_stencil = (int *) allocateGPU(nstencil * sizeof(int));
        memcpyToGPU(c_stencil, stencil, nstencil * sizeof(int));
    }

    if(c_binning.mbins == 0) {
        c_binning.mbins = mbins;
        c_binning.atoms_per_bin = atoms_per_bin;
        c_binning.bincount = (int *) allocateGPU(c_binning.mbins * sizeof(int));
        c_binning.bins = (int *) allocateGPU(c_binning.mbins * c_binning.atoms_per_bin * sizeof(int));
    }

    Neighbor_params np {
        .xprd = xprd,
        .yprd = yprd,
        .zprd = zprd,
        .bininvx = bininvx,
        .bininvy = bininvy,
        .bininvz = bininvz,
        .mbinxlo = mbinxlo,
        .mbinylo = mbinylo,
        .mbinzlo = mbinzlo,
        .nbinx = nbinx,
        .nbiny = nbiny,
        .nbinz = nbinz,
        .mbinx = mbinx,
        .mbiny = mbiny,
        .mbinz = mbinz
    };

    if(c_resize_needed == NULL) {
        c_resize_needed = (int *) allocateGPU(sizeof(int));
    }

    /* bin local & ghost atoms */
    binatoms_cuda(atom, &c_binning, c_resize_needed, &np, num_threads_per_block);
    if(c_new_maxneighs == NULL) {
        c_new_maxneighs = (int *) allocateGPU(sizeof(int));
    }

    int resize = 1;

    if(nall > nmax) {
        nmax = nall;
        d_neighbor->neighbors = (int *) reallocateGPU(d_neighbor->neighbors, nmax * neighbor->maxneighs * sizeof(int));
        d_neighbor->numneigh  = (int *) reallocateGPU(d_neighbor->numneigh,  nmax * sizeof(int));
    }

    /* loop over each atom, storing neighbors */
    while(resize) {
        resize = 0;
        memsetGPU(c_new_maxneighs, 0, sizeof(int));
        const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
        compute_neighborhood<<<num_blocks, num_threads_per_block>>>(atom->d_atom, *d_neighbor,
                                                                    np, atom->Nlocal, neighbor->maxneighs, nstencil, c_stencil,
                                                                    c_binning.bins, c_binning.atoms_per_bin, c_binning.bincount,
                                                                    c_new_maxneighs,
								                                    cutneighsq, atom->ntypes);

        cuda_assert("compute_neighborhood", hipPeekAtLastError());
        cuda_assert("compute_neighborhood", hipDeviceSynchronize());

        int new_maxneighs;
        memcpyFromGPU(&new_maxneighs, c_new_maxneighs, sizeof(int));
        if(new_maxneighs > neighbor->maxneighs){
            resize = 1;
        }

        if(resize) {
            printf("RESIZE %d\n", neighbor->maxneighs);
            neighbor->maxneighs = new_maxneighs * 1.2;
            printf("NEW SIZE %d\n", neighbor->maxneighs);
            neighbor->neighbors = (int *) reallocateGPU(neighbor->neighbors, atom->Nmax * neighbor->maxneighs * sizeof(int));
        }

    }

    hipProfilerStop();
}
