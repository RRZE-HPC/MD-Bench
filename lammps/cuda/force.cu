#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2021 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
//---
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
//---    
#include <likwid-marker.h>

extern "C" {

#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <allocate.h>
#include <neighbor.h>
#include <parameter.h>
#include <timing.h>
#include <util.h>

}

// cuda kernel
__global__ void calc_force(Atom a, MD_FLOAT cutforcesq, MD_FLOAT sigma6, MD_FLOAT epsilon, int Nlocal, int neigh_maxneighs, int *neigh_neighbors, int *neigh_numneigh) { 
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= Nlocal) {
        return;
    }

    Atom *atom = &a;

    const int numneighs = neigh_numneigh[i];

    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);

    MD_FLOAT fix = 0;
    MD_FLOAT fiy = 0;
    MD_FLOAT fiz = 0;

    for(int k = 0; k < numneighs; k++) {
        int j = neigh_neighbors[atom->Nlocal * k + i];
        MD_FLOAT delx = xtmp - atom_x(j);
        MD_FLOAT dely = ytmp - atom_y(j);
        MD_FLOAT delz = ztmp - atom_z(j);
        MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

#ifdef EXPLICIT_TYPES
        const int type_j = atom->type[j];
        const int type_ij = type_i * atom->ntypes + type_j;
        const MD_FLOAT cutforcesq = atom->cutforcesq[type_ij];
        const MD_FLOAT sigma6 = atom->sigma6[type_ij];
        const MD_FLOAT epsilon = atom->epsilon[type_ij];
#endif

        if(rsq < cutforcesq) {
            MD_FLOAT sr2 = 1.0 / rsq;
            MD_FLOAT sr6 = sr2 * sr2 * sr2 * sigma6;
            MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;
            fix += delx * force;
            fiy += dely * force;
            fiz += delz * force;
        }
    }

    atom_fx(i) = fix;
    atom_fy(i) = fiy;
    atom_fz(i) = fiz;
}

__global__ void kernel_initial_integrate(MD_FLOAT dtforce, MD_FLOAT dt, int Nlocal, Atom a) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= Nlocal ) {
        return;
    }

    Atom *atom = &a;

    atom_vx(i) += dtforce * atom_fx(i);
    atom_vy(i) += dtforce * atom_fy(i);
    atom_vz(i) += dtforce * atom_fz(i);
    atom_x(i) = atom_x(i) + dt * atom_vx(i);
    atom_y(i) = atom_y(i) + dt * atom_vy(i);
    atom_z(i) = atom_z(i) + dt * atom_vz(i);
}

__global__ void kernel_final_integrate(MD_FLOAT dtforce, int Nlocal, Atom a) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= Nlocal ) {
        return;
    }

    Atom *atom = &a;

    atom_vx(i) += dtforce * atom_fx(i);
    atom_vy(i) += dtforce * atom_fy(i);
    atom_vz(i) += dtforce * atom_fz(i);
}

extern "C" {

void finalIntegrate_cuda(bool reneigh, Parameter *param, Atom *atom, Atom *c_atom) {
    const int Nlocal = atom->Nlocal;
    const int num_threads_per_block = get_num_threads();
    const int num_blocks = ceil((float)Nlocal / (float)num_threads_per_block);

    kernel_final_integrate <<< num_blocks, num_threads_per_block >>> (param->dtforce, Nlocal, *c_atom);
    cuda_assert("kernel_final_integrate", hipPeekAtLastError());
    cuda_assert("kernel_final_integrate", hipDeviceSynchronize());

    if(reneigh) {
        memcpyFromGPU(atom->vx, c_atom->vx, sizeof(MD_FLOAT) * atom->Nlocal * 3);
    }
}

void initialIntegrate_cuda(bool reneigh, Parameter *param, Atom *atom, Atom *c_atom) {
    const int Nlocal = atom->Nlocal;
    const int num_threads_per_block = get_num_threads();
    const int num_blocks = ceil((float)Nlocal / (float)num_threads_per_block);

    kernel_initial_integrate <<< num_blocks, num_threads_per_block >>> (param->dtforce, param->dt, Nlocal, *c_atom);
    cuda_assert("kernel_initial_integrate", hipPeekAtLastError());
    cuda_assert("kernel_initial_integrate", hipDeviceSynchronize());

    if(reneigh) {
        memcpyFromGPU(atom->vx, c_atom->vx, sizeof(MD_FLOAT) * atom->Nlocal * 3);
    }
}

double computeForceLJFullNeigh_cuda(Parameter *param, Atom *atom, Neighbor *neighbor, Atom *c_atom, Neighbor *c_neighbor) {
    const int num_threads_per_block = get_num_threads();
    int Nlocal = atom->Nlocal;
#ifndef EXPLICIT_TYPES
    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6 = param->sigma6;
    MD_FLOAT epsilon = param->epsilon;
#endif

    /*
    int nDevices;
    hipGetDeviceCount(&nDevices);
    size_t free, total;
    for(int i = 0; i < nDevices; ++i) {
        hipMemGetInfo( &free, &total );
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("DEVICE %d/%d NAME: %s\r\n with %ld MB/%ld MB memory used", i + 1, nDevices, prop.name, free / 1024 / 1024, total / 1024 / 1024);
    }
    */


    // HINT: Run with cuda-memcheck ./MDBench-NVCC in case of error
    // checkCUDAError( "c_atom->fx memset", hipMemset(c_atom->fx, 0, sizeof(MD_FLOAT) * Nlocal * 3) );

    hipProfilerStart();
    const int num_blocks = ceil((float)Nlocal / (float)num_threads_per_block);
    double S = getTimeStamp();
    LIKWID_MARKER_START("force");

    calc_force <<< num_blocks, num_threads_per_block >>> (*c_atom, cutforcesq, sigma6, epsilon, Nlocal, neighbor->maxneighs, c_neighbor->neighbors, c_neighbor->numneigh);
    cuda_assert("calc_force", hipPeekAtLastError());
    cuda_assert("calc_force", hipDeviceSynchronize());
    hipProfilerStop();

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();
    return E-S;
}

}
