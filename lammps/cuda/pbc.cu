#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
//---

extern "C" {

#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <pbc.h>
#include <util.h>

}

extern int NmaxGhost;
extern int *PBCx, *PBCy, *PBCz;
static int c_NmaxGhost;
static int *c_PBCx, *c_PBCy, *c_PBCz;

__global__ void computeAtomsPbcUpdate(Atom a, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    Atom* atom = &a;
    if(i >= atom->Nlocal) {
        return;
    }

    if (atom_x(i) < 0.0) {
        atom_x(i) += xprd;
    } else if (atom_x(i) >= xprd) {
        atom_x(i) -= xprd;
    }

    if (atom_y(i) < 0.0) {
        atom_y(i) += yprd;
    } else if (atom_y(i) >= yprd) {
        atom_y(i) -= yprd;
    }

    if (atom_z(i) < 0.0) {
        atom_z(i) += zprd;
    } else if (atom_z(i) >= zprd) {
        atom_z(i) -= zprd;
    }
}

__global__ void computePbcUpdate(Atom a, int* PBCx, int* PBCy, int* PBCz, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nghost = a.Nghost;
    if(i >= Nghost) {
        return;
    }

    Atom* atom = &a;
    int *border_map = atom->border_map;
    int nlocal = atom->Nlocal;

    atom_x(nlocal + i) = atom_x(border_map[i]) + PBCx[i] * xprd;
    atom_y(nlocal + i) = atom_y(border_map[i]) + PBCy[i] * yprd;
    atom_z(nlocal + i) = atom_z(border_map[i]) + PBCz[i] * zprd;
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
void updatePbc_cuda(Atom *atom, Atom *c_atom, Parameter *param, bool doReneighbor) {
    const int num_threads_per_block = get_num_threads();

    if (doReneighbor) {
        c_atom->Natoms = atom->Natoms;
        c_atom->Nlocal = atom->Nlocal;
        c_atom->Nghost = atom->Nghost;
        c_atom->ntypes = atom->ntypes;

        if (atom->Nmax > c_atom->Nmax){ // the number of ghost atoms has increased -> more space is needed
            c_atom->Nmax = atom->Nmax;
            if(c_atom->x != NULL){ hipFree(c_atom->x); }
            if(c_atom->type != NULL){ hipFree(c_atom->type); }
            checkCUDAError( "updatePbc c_atom->x malloc", hipMalloc((void**)&(c_atom->x), sizeof(MD_FLOAT) * atom->Nmax * 3) );
            checkCUDAError( "updatePbc c_atom->type malloc", hipMalloc((void**)&(c_atom->type), sizeof(int) * atom->Nmax) );
        }
        // TODO if the sort is reactivated the atom->vx needs to be copied to GPU as well
        checkCUDAError( "updatePbc c_atom->x memcpy", hipMemcpy(c_atom->x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_atom->type memcpy", hipMemcpy(c_atom->type, atom->type, sizeof(int) * atom->Nmax, hipMemcpyHostToDevice) );

        if(c_NmaxGhost < NmaxGhost){
            c_NmaxGhost = NmaxGhost;
            if(c_PBCx != NULL){ hipFree(c_PBCx); }
            if(c_PBCy != NULL){ hipFree(c_PBCy); }
            if(c_PBCz != NULL){ hipFree(c_PBCz); }
            if(c_atom->border_map != NULL){ hipFree(c_atom->border_map); }
            checkCUDAError( "updatePbc c_PBCx malloc", hipMalloc((void**)&c_PBCx, NmaxGhost * sizeof(int)) );
            checkCUDAError( "updatePbc c_PBCy malloc", hipMalloc((void**)&c_PBCy, NmaxGhost * sizeof(int)) );
            checkCUDAError( "updatePbc c_PBCz malloc", hipMalloc((void**)&c_PBCz, NmaxGhost * sizeof(int)) );
            checkCUDAError( "updatePbc c_atom->border_map malloc", hipMalloc((void**)&(c_atom->border_map), NmaxGhost * sizeof(int)) );
        }
        checkCUDAError( "updatePbc c_PBCx memcpy", hipMemcpy(c_PBCx, PBCx, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_PBCy memcpy", hipMemcpy(c_PBCy, PBCy, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_PBCz memcpy", hipMemcpy(c_PBCz, PBCz, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_atom->border_map memcpy", hipMemcpy(c_atom->border_map, atom->border_map, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
    }

    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nghost / (float)num_threads_per_block);

    /*__global__ void computePbcUpdate(Atom a, int* PBCx, int* PBCy, int* PBCz,
     *                                                          MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd)
     * */
    computePbcUpdate<<<num_blocks, num_threads_per_block>>>(*c_atom, c_PBCx, c_PBCy, c_PBCz, xprd, yprd, zprd);
    checkCUDAError( "PeekAtLastError UpdatePbc", hipPeekAtLastError() );
    checkCUDAError( "DeviceSync UpdatePbc", hipDeviceSynchronize() );
}

void updateAtomsPbc_cuda(Atom* atom, Atom *c_atom, Parameter *param){
    const int num_threads_per_block = get_num_threads();
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
    /*void computeAtomsPbcUpdate(Atom a, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd)*/
    computeAtomsPbcUpdate<<<num_blocks, num_threads_per_block>>>(*c_atom, xprd, yprd, zprd);

    checkCUDAError( "PeekAtLastError UpdateAtomsPbc", hipPeekAtLastError() );
    checkCUDAError( "DeviceSync UpdateAtomsPbc", hipDeviceSynchronize() );
    checkCUDAError( "updateAtomsPbc position memcpy back", hipMemcpy(atom->x, c_atom->x, sizeof(MD_FLOAT) * atom->Nlocal * 3, hipMemcpyDeviceToHost) );
}
