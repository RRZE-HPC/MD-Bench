#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
//---

extern "C" {

#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <pbc.h>
#include <util.h>

}

extern int NmaxGhost;
extern int *PBCx, *PBCy, *PBCz;
static int c_NmaxGhost;
static int *c_PBCx, *c_PBCy, *c_PBCz;

__global__ void computeAtomsPbcUpdate(Atom a, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    Atom* atom = &a;
    if(i >= atom->Nlocal) {
        return;
    }

    if (atom_x(i) < 0.0) {
        atom_x(i) += xprd;
    } else if (atom_x(i) >= xprd) {
        atom_x(i) -= xprd;
    }

    if (atom_y(i) < 0.0) {
        atom_y(i) += yprd;
    } else if (atom_y(i) >= yprd) {
        atom_y(i) -= yprd;
    }

    if (atom_z(i) < 0.0) {
        atom_z(i) += zprd;
    } else if (atom_z(i) >= zprd) {
        atom_z(i) -= zprd;
    }
}

__global__ void computePbcUpdate(Atom a, int* PBCx, int* PBCy, int* PBCz, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nghost = a.Nghost;
    if(i >= Nghost) {
        return;
    }

    Atom* atom = &a;
    int *border_map = atom->border_map;
    int nlocal = atom->Nlocal;

    atom_x(nlocal + i) = atom_x(border_map[i]) + PBCx[i] * xprd;
    atom_y(nlocal + i) = atom_y(border_map[i]) + PBCy[i] * yprd;
    atom_z(nlocal + i) = atom_z(border_map[i]) + PBCz[i] * zprd;
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
void updatePbc_cuda(Atom *atom, Atom *c_atom, Parameter *param, bool doReneighbor) {
    const int num_threads_per_block = get_num_threads();

    if (doReneighbor) {
        c_atom->Natoms = atom->Natoms;
        c_atom->Nlocal = atom->Nlocal;
        c_atom->Nghost = atom->Nghost;
        c_atom->ntypes = atom->ntypes;

        if (atom->Nmax > c_atom->Nmax){ // the number of ghost atoms has increased -> more space is needed
            c_atom->Nmax = atom->Nmax;
            c_atom->x = (MD_FLOAT *) reallocateGPU(c_atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3);
            c_atom->type = (int *) reallocateGPU(c_atom->type, sizeof(int) * atom->Nmax);
        }

        memcpyToGPU(c_atom->x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3);
        memcpyToGPU(c_atom->type, atom->type, sizeof(int) * atom->Nmax);

        if(c_NmaxGhost < NmaxGhost) {
            c_NmaxGhost = NmaxGhost;
            c_PBCx = (int *) reallocateGPU(c_PBCx, NmaxGhost * sizeof(int));
            c_PBCy = (int *) reallocateGPU(c_PBCy, NmaxGhost * sizeof(int));
            c_PBCz = (int *) reallocateGPU(c_PBCz, NmaxGhost * sizeof(int));
            c_atom->border_map = (int *) reallocateGPU(c_atom->border_map, NmaxGhost * sizeof(int));
        }

        memcpyToGPU(c_PBCx, PBCx, NmaxGhost * sizeof(int));
        memcpyToGPU(c_PBCy, PBCy, NmaxGhost * sizeof(int));
        memcpyToGPU(c_PBCz, PBCz, NmaxGhost * sizeof(int));
        memcpyToGPU(c_atom->border_map, atom->border_map, NmaxGhost * sizeof(int));
    }

    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nghost / (float)num_threads_per_block);
    computePbcUpdate<<<num_blocks, num_threads_per_block>>>(*c_atom, c_PBCx, c_PBCy, c_PBCz, xprd, yprd, zprd);
    cuda_assert("computePbcUpdate", hipPeekAtLastError());
    cuda_assert("computePbcUpdate", hipDeviceSynchronize());
}

void updateAtomsPbc_cuda(Atom* atom, Atom *c_atom, Parameter *param) {
    const int num_threads_per_block = get_num_threads();
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
    computeAtomsPbcUpdate<<<num_blocks, num_threads_per_block>>>(*c_atom, xprd, yprd, zprd);
    cuda_assert("computeAtomsPbcUpdate", hipPeekAtLastError());
    cuda_assert("computeAtomsPbcUpdate", hipDeviceSynchronize());
    memcpyFromGPU(atom->x, c_atom->x, sizeof(MD_FLOAT) * atom->Nlocal * 3);
}
