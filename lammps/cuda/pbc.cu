#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 NHR@FAU, University Erlangen-Nuremberg.
 * All rights reserved. This file is part of MD-Bench.
 * Use of this source code is governed by a LGPL-3.0
 * license that can be found in the LICENSE file.
 */
#include <stdlib.h>
#include <stdio.h>
//---

extern "C" {

#include <allocate.h>
#include <atom.h>
#include <device.h>
#include <pbc.h>
#include <util.h>

}

extern int NmaxGhost;
extern int *PBCx, *PBCy, *PBCz;
static int c_NmaxGhost = 0;
static int *c_PBCx = NULL, *c_PBCy = NULL, *c_PBCz = NULL;

__global__ void computeAtomsPbcUpdate(DeviceAtom a, int nlocal, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    DeviceAtom *atom = &a;
    if(i >= nlocal) {
        return;
    }

    if (atom_x(i) < 0.0) {
        atom_x(i) += xprd;
    } else if (atom_x(i) >= xprd) {
        atom_x(i) -= xprd;
    }

    if (atom_y(i) < 0.0) {
        atom_y(i) += yprd;
    } else if (atom_y(i) >= yprd) {
        atom_y(i) -= yprd;
    }

    if (atom_z(i) < 0.0) {
        atom_z(i) += zprd;
    } else if (atom_z(i) >= zprd) {
        atom_z(i) -= zprd;
    }
}

__global__ void computePbcUpdate(DeviceAtom a, int nlocal, int nghost, int* PBCx, int* PBCy, int* PBCz, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= nghost) {
        return;
    }

    DeviceAtom* atom = &a;
    int *border_map = atom->border_map;
    atom_x(nlocal + i) = atom_x(border_map[i]) + PBCx[i] * xprd;
    atom_y(nlocal + i) = atom_y(border_map[i]) + PBCy[i] * yprd;
    atom_z(nlocal + i) = atom_z(border_map[i]) + PBCz[i] * zprd;
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
void updatePbc_cuda(Atom *atom, Parameter *param, bool reneigh) {
    const int num_threads_per_block = get_cuda_num_threads();

    if(reneigh) {
        memcpyToGPU(atom->d_atom.x,     atom->x,    sizeof(MD_FLOAT) * atom->Nmax * 3);
        memcpyToGPU(atom->d_atom.type,  atom->type, sizeof(int) * atom->Nmax);

        if(c_NmaxGhost < NmaxGhost) {
            c_NmaxGhost = NmaxGhost;
            c_PBCx = (int *) reallocateGPU(c_PBCx, NmaxGhost * sizeof(int));
            c_PBCy = (int *) reallocateGPU(c_PBCy, NmaxGhost * sizeof(int));
            c_PBCz = (int *) reallocateGPU(c_PBCz, NmaxGhost * sizeof(int));
            atom->d_atom.border_map = (int *) reallocateGPU(atom->d_atom.border_map, NmaxGhost * sizeof(int));
        }

        memcpyToGPU(c_PBCx, PBCx, NmaxGhost * sizeof(int));
        memcpyToGPU(c_PBCy, PBCy, NmaxGhost * sizeof(int));
        memcpyToGPU(c_PBCz, PBCz, NmaxGhost * sizeof(int));
        memcpyToGPU(atom->d_atom.border_map, atom->border_map, NmaxGhost * sizeof(int));
        cuda_assert("updatePbc.reneigh", hipPeekAtLastError());
        cuda_assert("updatePbc.reneigh", hipDeviceSynchronize());
    }

    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nghost / (float)num_threads_per_block);
    computePbcUpdate<<<num_blocks, num_threads_per_block>>>(atom->d_atom, atom->Nlocal, atom->Nghost, c_PBCx, c_PBCy, c_PBCz, xprd, yprd, zprd);
    cuda_assert("updatePbc", hipPeekAtLastError());
    cuda_assert("updatePbc", hipDeviceSynchronize());
}

void updateAtomsPbc_cuda(Atom* atom, Parameter *param) {
    const int num_threads_per_block = get_cuda_num_threads();
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
    computeAtomsPbcUpdate<<<num_blocks, num_threads_per_block>>>(atom->d_atom, atom->Nlocal, xprd, yprd, zprd);
    cuda_assert("computeAtomsPbcUpdate", hipPeekAtLastError());
    cuda_assert("computeAtomsPbcUpdate", hipDeviceSynchronize());
    memcpyFromGPU(atom->x, atom->d_atom.x, sizeof(MD_FLOAT) * atom->Nlocal * 3);
}
