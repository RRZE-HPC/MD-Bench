#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
//---

extern "C" {

#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <pbc.h>
#include <util.h>

}

extern int NmaxGhost;
extern int *PBCx, *PBCy, *PBCz;
static int c_NmaxGhost = 0;
static int *c_PBCx = NULL, *c_PBCy = NULL, *c_PBCz = NULL;

__global__ void computeAtomsPbcUpdate(DeviceAtom a, int nlocal, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    DeviceAtom *atom = &a;
    if(i >= nlocal) {
        return;
    }

    if (atom_x(i) < 0.0) {
        atom_x(i) += xprd;
    } else if (atom_x(i) >= xprd) {
        atom_x(i) -= xprd;
    }

    if (atom_y(i) < 0.0) {
        atom_y(i) += yprd;
    } else if (atom_y(i) >= yprd) {
        atom_y(i) -= yprd;
    }

    if (atom_z(i) < 0.0) {
        atom_z(i) += zprd;
    } else if (atom_z(i) >= zprd) {
        atom_z(i) -= zprd;
    }
}

__global__ void computePbcUpdate(DeviceAtom a, int nlocal, int nghost, int* PBCx, int* PBCy, int* PBCz, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= nghost) {
        return;
    }

    DeviceAtom* atom = &a;
    int *border_map = atom->border_map;
    atom_x(nlocal + i) = atom_x(border_map[i]) + PBCx[i] * xprd;
    atom_y(nlocal + i) = atom_y(border_map[i]) + PBCy[i] * yprd;
    atom_z(nlocal + i) = atom_z(border_map[i]) + PBCz[i] * zprd;
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
void updatePbc_cuda(Atom *atom, Parameter *param, bool reneigh) {
    const int num_threads_per_block = get_num_threads();

    if(reneigh) {
        memcpyToGPU(atom->d_atom.x,     atom->x,    sizeof(MD_FLOAT) * atom->Nmax * 3);
        memcpyToGPU(atom->d_atom.type,  atom->type, sizeof(int) * atom->Nmax);

        if(c_NmaxGhost < NmaxGhost) {
            c_NmaxGhost = NmaxGhost;
            c_PBCx = (int *) reallocateGPU(c_PBCx, NmaxGhost * sizeof(int));
            c_PBCy = (int *) reallocateGPU(c_PBCy, NmaxGhost * sizeof(int));
            c_PBCz = (int *) reallocateGPU(c_PBCz, NmaxGhost * sizeof(int));
            atom->d_atom.border_map = (int *) reallocateGPU(atom->d_atom.border_map, NmaxGhost * sizeof(int));
        }

        memcpyToGPU(c_PBCx, PBCx, NmaxGhost * sizeof(int));
        memcpyToGPU(c_PBCy, PBCy, NmaxGhost * sizeof(int));
        memcpyToGPU(c_PBCz, PBCz, NmaxGhost * sizeof(int));
        memcpyToGPU(atom->d_atom.border_map, atom->border_map, NmaxGhost * sizeof(int));
        cuda_assert("updatePbc.reneigh", hipPeekAtLastError());
        cuda_assert("updatePbc.reneigh", hipDeviceSynchronize());
    }

    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nghost / (float)num_threads_per_block);
    computePbcUpdate<<<num_blocks, num_threads_per_block>>>(atom->d_atom, atom->Nlocal, atom->Nghost, c_PBCx, c_PBCy, c_PBCz, xprd, yprd, zprd);
    cuda_assert("updatePbc", hipPeekAtLastError());
    cuda_assert("updatePbc", hipDeviceSynchronize());
}

void updateAtomsPbc_cuda(Atom* atom, Parameter *param) {
    const int num_threads_per_block = get_num_threads();
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
    computeAtomsPbcUpdate<<<num_blocks, num_threads_per_block>>>(atom->d_atom, atom->Nlocal, xprd, yprd, zprd);
    cuda_assert("computeAtomsPbcUpdate", hipPeekAtLastError());
    cuda_assert("computeAtomsPbcUpdate", hipDeviceSynchronize());
    memcpyFromGPU(atom->x, atom->d_atom.x, sizeof(MD_FLOAT) * atom->Nlocal * 3);
}
