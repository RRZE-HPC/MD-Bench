/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */

extern "C" {

#include <stdio.h>
#include <hip/hip_runtime.h>
//---
#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <neighbor.h>

void initCuda(Atom *atom, Neighbor *neighbor, Atom *c_atom, Neighbor *c_neighbor) {
    c_atom->Natoms = atom->Natoms;
    c_atom->Nlocal = atom->Nlocal;
    c_atom->Nghost = atom->Nghost;
    c_atom->Nmax = atom->Nmax;
    c_atom->ntypes = atom->ntypes;
    c_atom->border_map = NULL;

    c_atom->x               =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->Nmax * 3);
    c_atom->vx              =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->Nmax * 3);
    c_atom->fx              =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->Nmax * 3);
    c_atom->epsilon         =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    c_atom->sigma6          =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    c_atom->cutforcesq      =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    c_atom->type            =   (int *) allocateGPU(sizeof(int) * atom->Nmax * 3);
    c_neighbor->neighbors   =   (int *) allocateGPU(sizeof(int) * atom->Nmax * neighbor->maxneighs);
    c_neighbor->numneigh    =   (int *) allocateGPU(sizeof(int) * atom->Nmax);

    memcpyToGPU(c_atom->x,              atom->x,          sizeof(MD_FLOAT) * atom->Nmax * 3);
    memcpyToGPU(c_atom->vx,             atom->vx,         sizeof(MD_FLOAT) * atom->Nmax * 3);
    memcpyToGPU(c_atom->sigma6,         atom->sigma6,     sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    memcpyToGPU(c_atom->epsilon,        atom->epsilon,    sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    memcpyToGPU(c_atom->cutforcesq,     atom->cutforcesq, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    memcpyToGPU(c_atom->type,           atom->type,       sizeof(int) * atom->Nmax);
}

void cuda_assert(const char *label, hipError_t err) {
    if (err != hipSuccess) {
        printf("[CUDA Error]: %s: %s\r\n", label, hipGetErrorString(err));
        exit(-1);
    }
}

}
