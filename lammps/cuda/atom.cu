/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */

extern "C" {

#include <stdio.h>
#include <hip/hip_runtime.h>
//---
#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <neighbor.h>

void initCuda(Atom *atom, Neighbor *neighbor, Atom *c_atom, Neighbor *c_neighbor) {
    c_atom->Natoms = atom->Natoms;
    c_atom->Nlocal = atom->Nlocal;
    c_atom->Nghost = atom->Nghost;
    c_atom->Nmax = atom->Nmax;
    c_atom->ntypes = atom->ntypes;

    c_atom->border_map = NULL;

    const int Nlocal = atom->Nlocal;

    checkCUDAError( "c_atom->x malloc", hipMalloc((void**)&(c_atom->x), sizeof(MD_FLOAT) * atom->Nmax * 3) );
    checkCUDAError( "c_atom->x memcpy", hipMemcpy(c_atom->x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice) );

    checkCUDAError( "c_atom->fx malloc", hipMalloc((void**)&(c_atom->fx), sizeof(MD_FLOAT) * Nlocal * 3) );

    checkCUDAError( "c_atom->vx malloc", hipMalloc((void**)&(c_atom->vx), sizeof(MD_FLOAT) * Nlocal * 3) );
    checkCUDAError( "c_atom->vx memcpy", hipMemcpy(c_atom->vx, atom->vx, sizeof(MD_FLOAT) * Nlocal * 3, hipMemcpyHostToDevice) );

    checkCUDAError( "c_atom->type malloc", hipMalloc((void**)&(c_atom->type), sizeof(int) * atom->Nmax) );
    checkCUDAError( "c_atom->epsilon malloc", hipMalloc((void**)&(c_atom->epsilon), sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes) );
    checkCUDAError( "c_atom->sigma6 malloc", hipMalloc((void**)&(c_atom->sigma6), sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes) );
    checkCUDAError( "c_atom->cutforcesq malloc", hipMalloc((void**)&(c_atom->cutforcesq), sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes) );

    checkCUDAError( "c_neighbor->neighbors malloc", hipMalloc((void**)&c_neighbor->neighbors, sizeof(int) * Nlocal * neighbor->maxneighs) );
    checkCUDAError( "c_neighbor->numneigh malloc", hipMalloc((void**)&c_neighbor->numneigh, sizeof(int) * Nlocal) );

    checkCUDAError( "c_atom->type memcpy", hipMemcpy(c_atom->type, atom->type, sizeof(int) * atom->Nmax, hipMemcpyHostToDevice) );
    checkCUDAError( "c_atom->sigma6 memcpy", hipMemcpy(c_atom->sigma6, atom->sigma6, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice) );
    checkCUDAError( "c_atom->epsilon memcpy", hipMemcpy(c_atom->epsilon, atom->epsilon, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice) );

    checkCUDAError( "c_atom->cutforcesq memcpy", hipMemcpy(c_atom->cutforcesq, atom->cutforcesq, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice) );
}

void checkCUDAError(const char *msg, hipError_t err) {
    if (err != hipSuccess) {
        //print a human readable error message
        printf("[CUDA ERROR %s]: %s\r\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

}
