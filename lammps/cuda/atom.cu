/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */

extern "C" {

#include <stdio.h>
#include <hip/hip_runtime.h>
//---
#include <allocate.h>
#include <atom.h>
#include <cuda_atom.h>
#include <neighbor.h>

void initCuda(Atom *atom, Neighbor *neighbor) {
    DeviceAtom *d_atom = &(atom->d_atom);
    DeviceNeighbor *d_neighbor = &(neighbor->d_neighbor);

    d_atom->epsilon         =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    d_atom->sigma6          =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    d_atom->cutforcesq      =   (MD_FLOAT *) allocateGPU(sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    d_neighbor->neighbors   =   (int *) allocateGPU(sizeof(int) * atom->Nmax * neighbor->maxneighs);
    d_neighbor->numneigh    =   (int *) allocateGPU(sizeof(int) * atom->Nmax);

    memcpyToGPU(d_atom->x,              atom->x,          sizeof(MD_FLOAT) * atom->Nmax * 3);
    memcpyToGPU(d_atom->vx,             atom->vx,         sizeof(MD_FLOAT) * atom->Nmax * 3);
    memcpyToGPU(d_atom->sigma6,         atom->sigma6,     sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    memcpyToGPU(d_atom->epsilon,        atom->epsilon,    sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    memcpyToGPU(d_atom->cutforcesq,     atom->cutforcesq, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
    memcpyToGPU(d_atom->type,           atom->type,       sizeof(int) * atom->Nmax);
}

void cuda_assert(const char *label, hipError_t err) {
    if (err != hipSuccess) {
        printf("[CUDA Error]: %s: %s\r\n", label, hipGetErrorString(err));
        exit(-1);
    }
}

}
